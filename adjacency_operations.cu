#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include "float3_lib.cuh"

#include "LogPrint.h"

#define NUM_WORKGROUPS      64

#define uint32_t unsigned int
#define int32_t int

__global__
void getNumAdjacentClusters(
    uint32_t* paiRetNumAdjacentClusters,
    float3 const* paaVertexPositions,
    uint32_t const* paiBoundaryVertexIndices, 
    uint32_t const* paiNumBoundaryVertexIndices,
    uint32_t const* paiClusterFloat3Offsets,
    uint32_t iNumClusters)
{
    uint32_t iCluster = blockIdx.x * NUM_WORKGROUPS + threadIdx.x;
    if(iCluster >= iNumClusters)
    {
        return;
    }

    uint32_t iBoundaryVertexOffset = 0;
    for(uint32_t i = 0; i < iCluster; i++)
    {
        iBoundaryVertexOffset += paiNumBoundaryVertexIndices[i];
    }

    uint32_t iFloat3Offset = paiClusterFloat3Offsets[iCluster];
    uint32_t iNumBoundaryVertexIndices = paiNumBoundaryVertexIndices[iCluster];
    float3 const* aVertexPositions = paaVertexPositions + iFloat3Offset;
    uint32_t const* aiBoundaryVertexIndices = paiBoundaryVertexIndices + iBoundaryVertexOffset;

    //printf("cluster %d iNumBoundaryVertexIndices = %d iBoundaryVertexOffset = %d\n", 
    //    iCluster, 
    //    iNumBoundaryVertexIndices,
    //    iBoundaryVertexOffset);

    for(uint32_t i = 0; i < iNumBoundaryVertexIndices; i++)
    {
        uint32_t iVertexIndex = aiBoundaryVertexIndices[i];
        float3 const& vertexPosition = aVertexPositions[iVertexIndex];
        //if(iCluster == 1)
        //{
        //    printf("cluster1 %d (%.4f, %.4f, %.4f)\n",
        //        iVertexIndex,
        //        vertexPosition.x,
        //        vertexPosition.y,
        //        vertexPosition.z);
        //}

        for(uint32_t iCheckCluster = iCluster + 1; iCheckCluster < iNumClusters; iCheckCluster++)
        {
            if(iCheckCluster == iCluster)
            {
                continue;
            }

            uint32_t iCheckBoundaryVertexOffset = 0;
            for(uint32_t j = 0; j < iCheckCluster; j++)
            {
                iCheckBoundaryVertexOffset += paiNumBoundaryVertexIndices[j];
            }

            uint32_t iCheckFloat3Offset = paiClusterFloat3Offsets[iCheckCluster];
            uint32_t iNumCheckBoundaryVertexIndices = paiNumBoundaryVertexIndices[iCheckCluster];
            float3 const* aCheckVertexPositions = paaVertexPositions + iCheckFloat3Offset;
            uint32_t const* aiCheckBoundaryVertexIndices = paiBoundaryVertexIndices + iCheckBoundaryVertexOffset;

            for(uint32_t iCheckBoundaryVertex = 0; iCheckBoundaryVertex < iNumCheckBoundaryVertexIndices; iCheckBoundaryVertex++)
            {
                uint32_t iCheckVertexIndex = aiCheckBoundaryVertexIndices[iCheckBoundaryVertex];
                float3 const& checkVertexPosition = aCheckVertexPositions[iCheckVertexIndex];

                float fLength = lengthSquared(vertexPosition - checkVertexPosition);
                if(fLength <= 1.0e-8f)
                {
                    uint32_t iIndex = iCluster * iNumClusters + iCheckCluster;
                    uint32_t iCheckIndex = iCheckCluster * iNumClusters + iCluster;
                    paiRetNumAdjacentClusters[iIndex] += 1;
                    paiRetNumAdjacentClusters[iCheckIndex] += 1;
                    break;
                }
            }
            
        }   // for check cluster = 0 to num clusters
    
    }   // for i = 0 to num boundary vertex indices   

    if(iCluster % 100 == 0)
    {
        printf("cluster %d (%d) done\n", 
            iCluster,
            iNumClusters);
    }
}

#undef uint32_t
#undef int32_t

#include "adjacency_operations_cuda.h"

/*
**
*/
void buildClusterEdgeAdjacencyCUDA3(
    std::vector<std::vector<uint32_t>>& aaiNumAdjacentClusters,
    std::vector<std::vector<vec3>> const& aaVertexPositions,
    std::vector<std::vector<uint32_t>> const& aaiBoundaryVertexIndices)
{
    uint32_t iNumClusters = static_cast<uint32_t>(aaVertexPositions.size());

    // boundary vertex indices
    uint32_t* paiBoundaryVertexIndices = nullptr;
    uint32_t iNumTotalBoundaryVertexIndices = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        iNumTotalBoundaryVertexIndices += static_cast<uint32_t>(aaiBoundaryVertexIndices[iCluster].size());
    }
    hipMalloc(
        &paiBoundaryVertexIndices,
        iNumTotalBoundaryVertexIndices * sizeof(uint32_t));
    iNumTotalBoundaryVertexIndices = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        hipMemcpy(
            paiBoundaryVertexIndices + iNumTotalBoundaryVertexIndices,
            aaiBoundaryVertexIndices[iCluster].data(),
            aaiBoundaryVertexIndices[iCluster].size() * sizeof(uint32_t),
            hipMemcpyHostToDevice);
        iNumTotalBoundaryVertexIndices += static_cast<uint32_t>(aaiBoundaryVertexIndices[iCluster].size());
    }
    
    // num boundary vertex indices
    uint32_t* paiNumBoundaryVertexIndices = nullptr;
    hipMalloc(
        &paiNumBoundaryVertexIndices,
        iNumClusters * sizeof(uint32_t));
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        uint32_t iNumBoundaryVertices = static_cast<uint32_t>(aaiBoundaryVertexIndices[iCluster].size());
        hipMemcpy(
            paiNumBoundaryVertexIndices + iCluster,
            &iNumBoundaryVertices,
            sizeof(uint32_t),
            hipMemcpyHostToDevice);
    }

    // vertex positions
    uint32_t iNumTotalVertexPositions = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        iNumTotalVertexPositions += static_cast<uint32_t>(aaVertexPositions[iCluster].size());
    }
    float3* paaVertexPositions = nullptr;
    hipMalloc(
        &paaVertexPositions,
        iNumTotalVertexPositions * sizeof(float3));
    iNumTotalVertexPositions = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        hipMemcpy(
            paaVertexPositions + iNumTotalVertexPositions,
            aaVertexPositions[iCluster].data(),
            aaVertexPositions[iCluster].size() * sizeof(float3),
            hipMemcpyHostToDevice);
        iNumTotalVertexPositions += static_cast<uint32_t>(aaVertexPositions[iCluster].size());
    }

    // num cluster vertex positions
    uint32_t* paiRetNumAdjacentClusters = nullptr;
    hipMalloc(
        &paiRetNumAdjacentClusters,
        iNumClusters * iNumClusters * sizeof(uint32_t));

    // cluster array byte offset
    uint32_t* paiClusterFloat3Offsets = nullptr;
    hipMalloc(
        &paiClusterFloat3Offsets,
        iNumClusters * sizeof(uint32_t));
    uint32_t iOffsetFloat3 = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        hipMemcpy(
            paiClusterFloat3Offsets + iCluster,
            &iOffsetFloat3,
            sizeof(uint32_t),
            hipMemcpyHostToDevice);
        iOffsetFloat3 += static_cast<uint32_t>(aaVertexPositions[iCluster].size());
    }

    //uint32_t const kiTestCluster = 1;
    //for(uint32_t i = 0; i < aaiBoundaryVertexIndices[kiTestCluster].size(); i++)
    //{
    //    uint32_t iBoundaryVertexIndex = aaiBoundaryVertexIndices[kiTestCluster][i];
    //    vec3 const& vertexPosition = aaVertexPositions[kiTestCluster][iBoundaryVertexIndex];
    //    DEBUG_PRINTF("cluster%d %d (%.4f, %.4f, %.4f)\n",
    //        kiTestCluster,
    //        iBoundaryVertexIndex,
    //        vertexPosition.x,
    //        vertexPosition.y,
    //        vertexPosition.z);
    //}

    hipMemset(
        &paiRetNumAdjacentClusters,
        0,
        iNumClusters * iNumClusters * sizeof(uint32_t));

    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumClusters) / float(NUM_WORKGROUPS)));
    getNumAdjacentClusters<<<iNumBlocks, NUM_WORKGROUPS>>>(
        paiRetNumAdjacentClusters,
        paaVertexPositions,
        paiBoundaryVertexIndices,
        paiNumBoundaryVertexIndices,
        paiClusterFloat3Offsets,
        iNumClusters);

    aaiNumAdjacentClusters.resize(iNumClusters);
    for(uint32_t i = 0; i < iNumClusters; i++)
    {
        aaiNumAdjacentClusters[i].resize(iNumClusters);
        hipMemcpy(
            aaiNumAdjacentClusters[i].data(),
            paiRetNumAdjacentClusters + iNumClusters * i,
            iNumClusters * sizeof(uint32_t),
            hipMemcpyDeviceToHost);
    }

    hipFree(paiBoundaryVertexIndices);
    hipFree(paiNumBoundaryVertexIndices);
    hipFree(paaVertexPositions);
    hipFree(paiRetNumAdjacentClusters);
    hipFree(paiClusterFloat3Offsets);
    
}