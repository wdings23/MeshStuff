#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>

#define uint32_t unsigned int
#define int32_t int

#define MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP       3000

#define WORKGROUP_SIZE 64

/*
**
*/
__global__
void getSamePositions(
    float* pOutput,
    float* pData0,
    float* pData1,
    int iNumElements0,
    int iNumElements1)
{
    float fRet = 0.0f;
    for(int i = 0; i < iNumElements0; i++)
    {
        for(int j = 0; j < iNumElements1; j++)
        {
            if(pData0[i] == pData1[j])
            {
                fRet += 1.0f;
            }
        }
    }
    *pOutput = fRet;
}

struct TriAdjacentInfo
{
    uint32_t        miClusterGroup;
    uint32_t        miTriangle;
    uint32_t        miEdge;
    float3          mEdgePosition0;
    float3          mEdgePosition1;
};


/*
**
*/
__global__
void checkClusterGroupBoundaryVertices(
    uint32_t* aiRetClusterGroupBoundaryVertices,
    uint32_t* aiRetNumClusterGroupBoundaryVertices,
    uint32_t iNumClusterGroups,
    float* aaClusterGroupVertexPositions,
    uint32_t* aaiClusterClusterTrianglePositionIndices,
    uint32_t* aiNumClusterGroupVertexPositions,
    uint32_t* aiNumClusterGroupTrianglePositionIndices,
    uint32_t* aiClusterGroupVertexPositionOffsets,
    uint32_t* aiClusterGroupTriangleIndexOffsets)
{
    int iClusterGroup = blockIdx.x * 256 + threadIdx.x;
    if(iClusterGroup >= iNumClusterGroups)
    {
        return;
    }

    uint32_t iNumBoundaryVertices = 0;
    float* paClusterGroupTriangleVertexPositions = &aaClusterGroupVertexPositions[aiClusterGroupVertexPositionOffsets[iClusterGroup]];
    uint32_t* paiClusterGroupTriangleIndices = &aaiClusterClusterTrianglePositionIndices[aiClusterGroupTriangleIndexOffsets[iClusterGroup]];
    uint32_t iNumClusterGroupTriangleIndices = aiNumClusterGroupTrianglePositionIndices[iClusterGroup];

    for(uint32_t iTri = 0; iTri < iNumClusterGroupTriangleIndices; iTri += 3)
    {
        uint32_t aiAdjacentEdges[3] = { 0, 0, 0 };

        for(uint32_t iCheckClusterGroup = 0; iCheckClusterGroup < iNumClusterGroups; iCheckClusterGroup++)
        {
            uint32_t iNumCheckClusterGroupTriangleIndices = aiNumClusterGroupTrianglePositionIndices[iCheckClusterGroup];
            float* paCheckClusterGroupTriangleVertexPositions = &aaClusterGroupVertexPositions[aiClusterGroupVertexPositionOffsets[iCheckClusterGroup]];
            uint32_t* paiCheckClusterGroupTriangleIndices = &aaiClusterClusterTrianglePositionIndices[aiClusterGroupTriangleIndexOffsets[iCheckClusterGroup]];
            
            for(uint32_t iCheckTri = 0; iCheckTri < iNumCheckClusterGroupTriangleIndices; iCheckTri += 3)
            {
                if(iClusterGroup == iCheckClusterGroup && iTri == iCheckTri)
                {
                    continue;
                }

                // check the number of same vertex positions
                uint32_t aiSamePositionIndices[3];
                aiSamePositionIndices[0] = 0; aiSamePositionIndices[1] = 0; aiSamePositionIndices[2] = 0;
                uint32_t iNumSamePositions = 0;
                for(uint32_t i = 0; i < 3; i++)
                {
                    uint32_t iPos = paiClusterGroupTriangleIndices[iTri + i];
                    float fX = paClusterGroupTriangleVertexPositions[iPos * 3];
                    float fY = paClusterGroupTriangleVertexPositions[iPos * 3 + 1];
                    float fZ = paClusterGroupTriangleVertexPositions[iPos * 3 + 2];

                    for(uint32_t j = 0; j < 3; j++)
                    {
                        uint32_t iCheckPos = paiCheckClusterGroupTriangleIndices[iCheckTri + j];
                        float fCheckX = paCheckClusterGroupTriangleVertexPositions[iCheckPos * 3];
                        float fCheckY = paCheckClusterGroupTriangleVertexPositions[iCheckPos * 3 + 1];
                        float fCheckZ = paCheckClusterGroupTriangleVertexPositions[iCheckPos * 3 + 2];
                        
                        float fDiffX = fCheckX - fX;
                        float fDiffY = fCheckY - fY;
                        float fDiffZ = fCheckZ - fZ;

                        if((fDiffX * fDiffX + fDiffY * fDiffY + fDiffZ * fDiffZ) < 1.0e-6f)
                        {
                            aiSamePositionIndices[i] = 1;
                            ++iNumSamePositions;
                            break;
                        }
                    }
                }   // for i = 0 to 3, checking same position

                if(iNumSamePositions >= 2)
                {
                    // edge index based on the same vertex positions
                    uint32_t iEdge = 0xffffffff;
                    if(aiSamePositionIndices[0] == 1 && aiSamePositionIndices[1] == 1)
                    {
                        iEdge = 0;
                    }
                    else if(aiSamePositionIndices[0] == 1 && aiSamePositionIndices[2] == 1)
                    {
                        iEdge = 1;
                    }
                    else if(aiSamePositionIndices[1] == 1 && aiSamePositionIndices[2] == 1)
                    {
                        iEdge = 2;
                    }

                    if(iEdge <= 2)
                    {
                        aiAdjacentEdges[iEdge] = 1;
                    }
                }

            }   // for check tri = 0 to num triangles
        
        }   // for check cluster group = 0 to num check cluster groups
    
        if(aiAdjacentEdges[0] == 0)
        {
            uint32_t iIndex = iClusterGroup * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP + iNumBoundaryVertices;
            aiRetClusterGroupBoundaryVertices[iIndex] = paiClusterGroupTriangleIndices[iTri];
            aiRetClusterGroupBoundaryVertices[iIndex + 1] = paiClusterGroupTriangleIndices[iTri+1];
            iNumBoundaryVertices += 2;
        }
        else if(aiAdjacentEdges[1] == 0)
        {
            uint32_t iIndex = iClusterGroup * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP + iNumBoundaryVertices;
            aiRetClusterGroupBoundaryVertices[iIndex] = paiClusterGroupTriangleIndices[iTri];
            aiRetClusterGroupBoundaryVertices[iIndex + 1] = paiClusterGroupTriangleIndices[iTri + 2];
            iNumBoundaryVertices += 2;
        }
        else if(aiAdjacentEdges[2] == 0)
        {
            uint32_t iIndex = iClusterGroup * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP + iNumBoundaryVertices;
            aiRetClusterGroupBoundaryVertices[iIndex] = paiClusterGroupTriangleIndices[iTri + 1];
            aiRetClusterGroupBoundaryVertices[iIndex + 1] = paiClusterGroupTriangleIndices[iTri + 2];
            iNumBoundaryVertices += 2;
        }

    }   // for tri = 0 to num triangles

    aiRetNumClusterGroupBoundaryVertices[iClusterGroup] = iNumBoundaryVertices;
}


/*
**
*/
__device__
void getTriangleIndexFromThreadIndex(
    uint32_t* piRetClusterGroup,
    uint32_t* piRetTri,
    uint32_t iThreadIndex,
    uint32_t iNumClusterGroups,
    uint32_t* aiNumClusterGroupVertexPositions)
{
    uint32_t iClusterGroup = 0;
    uint32_t iTotalTris = 0;
    for(iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        if(iTotalTris + aiNumClusterGroupVertexPositions[iClusterGroup] > iThreadIndex)
        {
            break;
        }

        iTotalTris += aiNumClusterGroupVertexPositions[iClusterGroup];
    }

    *piRetClusterGroup = iClusterGroup;
    *piRetTri = (iThreadIndex >= aiNumClusterGroupVertexPositions[0]) ? iThreadIndex - iTotalTris : iThreadIndex;
}

/*
**
*/
__global__
void checkClusterGroupBoundaryVertices2(
    uint32_t* aiRetClusterGroupBoundaryVertices,
    uint32_t* aiRetNumClusterGroupBoundaryVertices,
    uint32_t iNumClusterGroups,
    float* aaClusterGroupVertexPositions,
    uint32_t* aaiClusterClusterTrianglePositionIndices,
    uint32_t* aiNumClusterGroupVertexPositions,
    uint32_t* aiNumClusterGroupTrianglePositionIndices,
    uint32_t* aiClusterGroupVertexPositionOffsets,
    uint32_t* aiClusterGroupTriangleIndexOffsets,
    uint32_t iNumTotalTriangleIndices)
{
    uint32_t iClusterGroup = 0;
    uint32_t iTri = 0;
    uint32_t iThreadIndex = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    getTriangleIndexFromThreadIndex(
        &iClusterGroup,
        &iTri,
        iThreadIndex * 3,
        iNumClusterGroups,
        aiNumClusterGroupTrianglePositionIndices);

    if(iTri >= iNumTotalTriangleIndices)
    {
        return;
    }

    //uint32_t iNumBoundaryVertices = 0;
    float* paClusterGroupTriangleVertexPositions = &aaClusterGroupVertexPositions[aiClusterGroupVertexPositionOffsets[iClusterGroup]];
    uint32_t* paiClusterGroupTriangleIndices = &aaiClusterClusterTrianglePositionIndices[aiClusterGroupTriangleIndexOffsets[iClusterGroup]];
    //uint32_t iNumClusterGroupTriangleIndices = aiNumClusterGroupTrianglePositionIndices[iClusterGroup];

    //for(uint32_t iTri = 0; iTri < iNumClusterGroupTriangleIndices; iTri += 3)
    {
        uint32_t aiAdjacentEdges[3] = { 0, 0, 0 };

        for(uint32_t iCheckClusterGroup = 0; iCheckClusterGroup < iNumClusterGroups; iCheckClusterGroup++)
        {
            uint32_t iNumCheckClusterGroupTriangleIndices = aiNumClusterGroupTrianglePositionIndices[iCheckClusterGroup];
            float* paCheckClusterGroupTriangleVertexPositions = &aaClusterGroupVertexPositions[aiClusterGroupVertexPositionOffsets[iCheckClusterGroup]];
            uint32_t* paiCheckClusterGroupTriangleIndices = &aaiClusterClusterTrianglePositionIndices[aiClusterGroupTriangleIndexOffsets[iCheckClusterGroup]];

            for(uint32_t iCheckTri = 0; iCheckTri < iNumCheckClusterGroupTriangleIndices; iCheckTri += 3)
            {
                if(iClusterGroup == iCheckClusterGroup && iTri == iCheckTri)
                {
                    continue;
                }

                // check the number of same vertex positions
                uint32_t aiSamePositionIndices[3];
                aiSamePositionIndices[0] = 0; aiSamePositionIndices[1] = 0; aiSamePositionIndices[2] = 0;
                uint32_t iNumSamePositions = 0;
                for(uint32_t i = 0; i < 3; i++)
                {
                    uint32_t iPos = paiClusterGroupTriangleIndices[iTri + i];
                    float fX = paClusterGroupTriangleVertexPositions[iPos * 3];
                    float fY = paClusterGroupTriangleVertexPositions[iPos * 3 + 1];
                    float fZ = paClusterGroupTriangleVertexPositions[iPos * 3 + 2];

                    for(uint32_t j = 0; j < 3; j++)
                    {
                        uint32_t iCheckPos = paiCheckClusterGroupTriangleIndices[iCheckTri + j];
                        float fCheckX = paCheckClusterGroupTriangleVertexPositions[iCheckPos * 3];
                        float fCheckY = paCheckClusterGroupTriangleVertexPositions[iCheckPos * 3 + 1];
                        float fCheckZ = paCheckClusterGroupTriangleVertexPositions[iCheckPos * 3 + 2];

                        float fDiffX = fCheckX - fX;
                        float fDiffY = fCheckY - fY;
                        float fDiffZ = fCheckZ - fZ;

                        if((fDiffX * fDiffX + fDiffY * fDiffY + fDiffZ * fDiffZ) < 1.0e-6f)
                        {
                            aiSamePositionIndices[i] = 1;
                            ++iNumSamePositions;
                            break;
                        }
                    }
                }   // for i = 0 to 3, checking same position

                if(iNumSamePositions >= 2)
                {
                    // edge index based on the same vertex positions
                    uint32_t iEdge = 0xffffffff;
                    if(aiSamePositionIndices[0] == 1 && aiSamePositionIndices[1] == 1)
                    {
                        iEdge = 0;
                    }
                    else if(aiSamePositionIndices[0] == 1 && aiSamePositionIndices[2] == 1)
                    {
                        iEdge = 1;
                    }
                    else if(aiSamePositionIndices[1] == 1 && aiSamePositionIndices[2] == 1)
                    {
                        iEdge = 2;
                    }

                    if(iEdge <= 2)
                    {
                        aiAdjacentEdges[iEdge] = 1;
                    }
                }

            }   // for check tri = 0 to num triangles

        }   // for check cluster group = 0 to num check cluster groups

        if(aiAdjacentEdges[0] == 0)
        {
            uint32_t iNumBoundaryVertices = atomicAdd(&aiRetNumClusterGroupBoundaryVertices[iClusterGroup], 2);
            uint32_t iIndex = iClusterGroup * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP + iNumBoundaryVertices;
            aiRetClusterGroupBoundaryVertices[iIndex] = paiClusterGroupTriangleIndices[iTri];
            aiRetClusterGroupBoundaryVertices[iIndex + 1] = paiClusterGroupTriangleIndices[iTri + 1];
        }
        else if(aiAdjacentEdges[1] == 0)
        {
            uint32_t iNumBoundaryVertices = atomicAdd(&aiRetNumClusterGroupBoundaryVertices[iClusterGroup], 2);
            uint32_t iIndex = iClusterGroup * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP + iNumBoundaryVertices;
            aiRetClusterGroupBoundaryVertices[iIndex] = paiClusterGroupTriangleIndices[iTri];
            aiRetClusterGroupBoundaryVertices[iIndex + 1] = paiClusterGroupTriangleIndices[iTri + 2];
        }
        else if(aiAdjacentEdges[2] == 0)
        {
            uint32_t iNumBoundaryVertices = atomicAdd(&aiRetNumClusterGroupBoundaryVertices[iClusterGroup], 2);
            uint32_t iIndex = iClusterGroup * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP + iNumBoundaryVertices;
            aiRetClusterGroupBoundaryVertices[iIndex] = paiClusterGroupTriangleIndices[iTri + 1];
            aiRetClusterGroupBoundaryVertices[iIndex + 1] = paiClusterGroupTriangleIndices[iTri + 2];
        }

    }   // for tri = 0 to num triangles
}

/*
**
*/
__global__
void checkClusterAdjacency(
    uint32_t* aiNumAdjacentVertices,
    float* afTotalClusterVertexPositionComponents,
    uint32_t* aiNumVertexPositionComponents,
    uint32_t* aiClusterVertexPositionComponentOffsets,
    uint32_t iNumTotalClusters,
    bool bOnlyEdgeAdjacent)
{
    uint32_t iCluster = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iCluster >= iNumTotalClusters)
    {
        return;
    }

    float* aClusterVertexPositionComponents = &afTotalClusterVertexPositionComponents[aiClusterVertexPositionComponentOffsets[iCluster]];
    uint32_t iNumVertexPositionComponents = aiNumVertexPositionComponents[iCluster];

    for(uint32_t iCheckCluster = 0; iCheckCluster < iNumTotalClusters; iCheckCluster++)
    {
        if(iCheckCluster == iCluster)
        {
            continue;
        }

        uint32_t iNumAdjacentVertices = 0;
        float* aCheckClusterVertexPositionComponents = &afTotalClusterVertexPositionComponents[aiClusterVertexPositionComponentOffsets[iCheckCluster]];
        uint32_t iNumCheckVertexPositionComponents = aiNumVertexPositionComponents[iCheckCluster];

        for(uint32_t iVertComponent = 0; iVertComponent < iNumVertexPositionComponents; iVertComponent += 3)
        {
            float fX = aClusterVertexPositionComponents[iVertComponent];
            float fY = aClusterVertexPositionComponents[iVertComponent + 1];
            float fZ = aClusterVertexPositionComponents[iVertComponent + 2];

            for(uint32_t iCheckVertComponent = 0; iCheckVertComponent < iNumCheckVertexPositionComponents; iCheckVertComponent += 3)
            {
                float fCheckX = aCheckClusterVertexPositionComponents[iCheckVertComponent];
                float fCheckY = aCheckClusterVertexPositionComponents[iCheckVertComponent + 1];
                float fCheckZ = aCheckClusterVertexPositionComponents[iCheckVertComponent + 2];

                float fDiffX = fX - fCheckX;
                float fDiffY = fY - fCheckY;
                float fDiffZ = fZ - fCheckZ;

                float fLength = fDiffX * fDiffX + fDiffY * fDiffY + fDiffZ * fDiffZ;
                if(fLength <= 1.0e-8f)
                {
                    if(bOnlyEdgeAdjacent && iNumAdjacentVertices >= 2)
                    {
                        break;
                    }
                    else
                    {
                        ++iNumAdjacentVertices;
                        break;
                    }
                    
                }
            }
        }

        uint32_t iIndex = iCluster * iNumTotalClusters + iCheckCluster;
        aiNumAdjacentVertices[iIndex] = iNumAdjacentVertices;
    }
}

/*
**
*/
__device__
void getClusterGroupAndVertexIndex(
    uint32_t* iRetClusterGroup,
    uint32_t* iRetVertexComponentIndex,
    uint32_t iTotalVertexIndex,
    uint32_t* aiNumVertexPositionComponents,
    uint32_t iNumClusterGroups)
{
    uint32_t iTotalVertexComponentIndex = iTotalVertexIndex * 3;
    uint32_t iTotalVertexPositionComponents = 0;
    uint32_t iClusterGroup = 0;
    for(iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        if(iTotalVertexPositionComponents + aiNumVertexPositionComponents[iClusterGroup] > iTotalVertexComponentIndex)
        {
            break;
        }

        iTotalVertexPositionComponents += aiNumVertexPositionComponents[iClusterGroup];
    }

    *iRetClusterGroup = iClusterGroup;
    *iRetVertexComponentIndex = iTotalVertexComponentIndex - iTotalVertexPositionComponents;
}

/*
**
*/
__global__
void checkClusterGroupAdjacency(
    uint32_t* aiAdjacentClusterGroupVertexIndices,
    uint32_t* aiNumAdjacentClusterGroupVertices,
    float* afTotalClusterGroupVertexPositionComponents,
    uint32_t* aiNumVertexPositionComponents,
    uint32_t* aiClusterGroupVertexPositionComponentOffsets,
    uint32_t iNumTotalVertexIndices,
    uint32_t iNumTotalClusterGroups)
{
    uint32_t iTotalVertexIndex = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iTotalVertexIndex >= iNumTotalVertexIndices / 3)
    {
        return;
    }

    uint32_t iClusterGroup = 0, iVertComponent = 0;
    getClusterGroupAndVertexIndex(
        &iClusterGroup,
        &iVertComponent,
        iTotalVertexIndex,
        aiNumVertexPositionComponents,
        iNumTotalClusterGroups);

if(iClusterGroup == 385 && iVertComponent >= 870 * 3)
{
    getClusterGroupAndVertexIndex(
        &iClusterGroup,
        &iVertComponent,
        iTotalVertexIndex,
        aiNumVertexPositionComponents,
        iNumTotalClusterGroups);

    printf("wtf\n");
}

    uint32_t iClusterGroupVertexPositionComponentOffset = aiClusterGroupVertexPositionComponentOffsets[iClusterGroup];
    float* aClusterVertexPositionComponents = &afTotalClusterGroupVertexPositionComponents[iClusterGroupVertexPositionComponentOffset];
//    uint32_t iNumVertexPositionComponents = aiNumVertexPositionComponents[iClusterGroup];

    float fX = aClusterVertexPositionComponents[iVertComponent];
    float fY = aClusterVertexPositionComponents[iVertComponent + 1];
    float fZ = aClusterVertexPositionComponents[iVertComponent + 2];

    for(uint32_t iCheckClusterGroup = 0; iCheckClusterGroup < iNumTotalClusterGroups; iCheckClusterGroup++)
    {
        if(iCheckClusterGroup == iClusterGroup)
        {
            continue;
        }

        uint32_t iCheckClusterGroupVertexPositionComponentOffset = aiClusterGroupVertexPositionComponentOffsets[iCheckClusterGroup];
        float* aCheckClusterVertexPositionComponents = &afTotalClusterGroupVertexPositionComponents[iCheckClusterGroupVertexPositionComponentOffset];
        uint32_t iNumCheckVertexPositionComponents = aiNumVertexPositionComponents[iCheckClusterGroup];

        for(uint32_t iCheckVertComponent = 0; iCheckVertComponent < iNumCheckVertexPositionComponents; iCheckVertComponent += 3)
        {
            float fCheckX = aCheckClusterVertexPositionComponents[iCheckVertComponent];
            float fCheckY = aCheckClusterVertexPositionComponents[iCheckVertComponent + 1];
            float fCheckZ = aCheckClusterVertexPositionComponents[iCheckVertComponent + 2];

            float fDiffX = fX - fCheckX;
            float fDiffY = fY - fCheckY;
            float fDiffZ = fZ - fCheckZ;

            float fLength = fDiffX * fDiffX + fDiffY * fDiffY + fDiffZ * fDiffZ;
            if(fLength <= 1.0e-10f)
            {
                uint32_t iArrayIndex = atomicAdd(&aiNumAdjacentClusterGroupVertices[iClusterGroup], 1);
if(iArrayIndex >= MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP)
{
    printf("wtf\n");
}
if(iClusterGroup == 385)
{
    printf("debug\n");
}

                uint32_t iIndex = iClusterGroup * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP + iArrayIndex;
                aiAdjacentClusterGroupVertexIndices[iIndex] = iVertComponent / 3;
            }
            
        }   // for check vertex component = 0 to num vertex components

    }   // for check cluster group = 0 to num cluster groups 
}

/*
**
*/
__global__
void computeEdgeCollapseInfo(
    float* afRetEdgeCollapseCosts,
    uint32_t* aiRetEdgeCollapseVertexIndices0,
    uint32_t* aiRetEdgeCollapseVertexIndices1,
    float* afRetEdgeCollapseVertexPositions,
    float* afRetEdgeCollapseVertexNormals,
    float* afRetEdgeCollapseVertexUVs,
    uint32_t* aiClusterGroupNonBoundaryVertexIndices,
    float* afVertexPositionComponents,
    float* afVertexNormalComponents,
    float* afVertexUVComponents,
    float* afQuadrics,
    float* afVertexNormalPlaneAngles,
    uint32_t* aiClusterGroupEdgePairs,
    uint32_t* aiClusterGroupTrianglePositionIndicesGPU,
    uint32_t* aiClusterGroupTriangleNormalIndicesGPU,
    uint32_t* aiClusterGroupTriangleUVIndicesGPU,
    uint32_t* aiNormalIndexToEdgeMap,
    uint32_t* aiUVIndexToEdgeMap,
    uint32_t iNumClusterGroupTrianglePositionIndices,
    uint32_t iNumClusterGroupNonBoundaryVertices,
    uint32_t iNumEdges)
{
    uint32_t iEdge = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iEdge >= iNumEdges)
    {
        return;
    }

    uint32_t iEdgeIndex = iEdge * 2;
    uint32_t iEdgePos0 = aiClusterGroupEdgePairs[iEdgeIndex];
    uint32_t iEdgePos1 = aiClusterGroupEdgePairs[iEdgeIndex + 1];

    uint32_t iNorm0 = aiNormalIndexToEdgeMap[iEdgeIndex];
    uint32_t iNorm1 = aiNormalIndexToEdgeMap[iEdgeIndex + 1];

    uint32_t iUV0 = aiUVIndexToEdgeMap[iEdgeIndex];
    uint32_t iUV1 = aiUVIndexToEdgeMap[iEdgeIndex + 1];

    aiRetEdgeCollapseVertexIndices0[iEdge] = iEdgePos0;
    aiRetEdgeCollapseVertexIndices1[iEdge] = iEdgePos1;

    // check which edge vertices are non-boundary
    bool bValid0 = false;
    bool bValid1 = false;
    for(uint32_t i = 0; i < iNumClusterGroupNonBoundaryVertices; i++)
    {
        if(aiClusterGroupNonBoundaryVertexIndices[i] == iEdgePos0)
        {
            bValid0 = true;
        }

        if(aiClusterGroupNonBoundaryVertexIndices[i] == iEdgePos1)
        {
            bValid1 = true;
        }

        if(bValid0 && bValid1)
        {
            break;
        }
    }

    if(!bValid0 && !bValid0)
    {
        afRetEdgeCollapseCosts[iEdge] = 1.0e+10f;
        aiRetEdgeCollapseVertexIndices0[iEdge] = 0;
        aiRetEdgeCollapseVertexIndices1[iEdge] = 0;
        return;
    }

    float afQuadrics0[16];
    memcpy(afQuadrics0, &afQuadrics[iEdgePos0 * 16], sizeof(float) * 16);

    float afQuadrics1[16];
    memcpy(afQuadrics1, &afQuadrics[iEdgePos1 * 16], sizeof(float) * 16);

    // normal plane angles for feature value
    float fTotalNormalPlaneAngles0 = afVertexNormalPlaneAngles[iEdgePos0];
    float fTotalNormalPlaneAngles1 = afVertexNormalPlaneAngles[iEdgePos1];

    // feature value
//    float const kfFeatureMult = 1.0f;
    float fDiffX = afVertexPositionComponents[iEdgePos1 * 3] - afVertexPositionComponents[iEdgePos0 * 3];
    float fDiffY = afVertexPositionComponents[iEdgePos1 * 3 + 1] - afVertexPositionComponents[iEdgePos0 * 3 + 1];
    float fDiffZ = afVertexPositionComponents[iEdgePos1 * 3 + 2] - afVertexPositionComponents[iEdgePos0 * 3 + 2];
    float fEdgeLength = fDiffX * fDiffX + fDiffY * fDiffY + fDiffZ * fDiffZ;
    float fFeatureValue = fEdgeLength * (1.0f + 0.5f * (fTotalNormalPlaneAngles0 + fTotalNormalPlaneAngles1));

    float afEdgeQuadrics[16];
    for(uint32_t i = 0; i < 16; i++)
    {
        afEdgeQuadrics[i] = afQuadrics0[i] + afQuadrics1[i];
    }
    afEdgeQuadrics[15] += fFeatureValue;

    //if(iEdgePos0 == 0 && iEdgePos1 == 1)
    //{
    //    printf("\n\n*********\n\n");
    //    for(uint32_t i = 0; i < 16; i++)
    //    {
    //        printf("%.4f\n", afEdgeQuadrics[i]);
    //    }
    //
    //    printf("****\n");
    //}

    if(bValid0 == false)
    {
        // boundary
        afRetEdgeCollapseVertexPositions[iEdge * 3] =     afVertexPositionComponents[iEdgePos0 * 3];
        afRetEdgeCollapseVertexPositions[iEdge * 3 + 1] = afVertexPositionComponents[iEdgePos0 * 3 + 1];
        afRetEdgeCollapseVertexPositions[iEdge * 3 + 2] = afVertexPositionComponents[iEdgePos0 * 3 + 2];

        afRetEdgeCollapseVertexNormals[iEdge * 3] =     afVertexNormalComponents[iNorm0 * 3];
        afRetEdgeCollapseVertexNormals[iEdge * 3 + 1] = afVertexNormalComponents[iNorm0 * 3 + 1];
        afRetEdgeCollapseVertexNormals[iEdge * 3 + 2] = afVertexNormalComponents[iNorm0 * 3 + 2];

        afRetEdgeCollapseVertexUVs[iEdge * 3] =     afVertexUVComponents[iUV0 * 3];
        afRetEdgeCollapseVertexUVs[iEdge * 3 + 1] = afVertexUVComponents[iUV0 * 3 + 1];
    }
    else if(bValid1 == false)
    {
        // boundary
        afRetEdgeCollapseVertexPositions[iEdge * 3] = afVertexPositionComponents[iEdgePos1 * 3];
        afRetEdgeCollapseVertexPositions[iEdge * 3 + 1] = afVertexPositionComponents[iEdgePos1 * 3 + 1];
        afRetEdgeCollapseVertexPositions[iEdge * 3 + 2] = afVertexPositionComponents[iEdgePos1 * 3 + 2];

        afRetEdgeCollapseVertexNormals[iEdge * 3] =     afVertexNormalComponents[iNorm1 * 3];
        afRetEdgeCollapseVertexNormals[iEdge * 3 + 1] = afVertexNormalComponents[iNorm1 * 3 + 1];
        afRetEdgeCollapseVertexNormals[iEdge * 3 + 2] = afVertexNormalComponents[iNorm1 * 3 + 2];

        afRetEdgeCollapseVertexUVs[iEdge * 3] =     afVertexUVComponents[iUV1 * 3];
        afRetEdgeCollapseVertexUVs[iEdge * 3 + 1] = afVertexUVComponents[iUV1 * 3 + 1];
    }
    else
    {
        // mid point
        afRetEdgeCollapseVertexPositions[iEdge * 3] =     (afVertexPositionComponents[iEdgePos0 * 3]     + afVertexPositionComponents[iEdgePos1 * 3]) * 0.5f;
        afRetEdgeCollapseVertexPositions[iEdge * 3 + 1] = (afVertexPositionComponents[iEdgePos0 * 3 + 1] + afVertexPositionComponents[iEdgePos1 * 3 + 1]) * 0.5f;
        afRetEdgeCollapseVertexPositions[iEdge * 3 + 2] = (afVertexPositionComponents[iEdgePos0 * 3 + 2] + afVertexPositionComponents[iEdgePos1 * 3 + 2]) * 0.5f;

        afRetEdgeCollapseVertexNormals[iEdge * 3] =     (afVertexNormalComponents[iNorm0 * 3]     + afVertexNormalComponents[iNorm1 * 3]) * 0.5f;
        afRetEdgeCollapseVertexNormals[iEdge * 3 + 1] = (afVertexNormalComponents[iNorm0 * 3 + 1] + afVertexNormalComponents[iNorm1 * 3 + 1]) * 0.5f;
        afRetEdgeCollapseVertexNormals[iEdge * 3 + 2] = (afVertexNormalComponents[iNorm0 * 3 + 2] + afVertexNormalComponents[iNorm1 * 3 + 2]) * 0.5f;

        afRetEdgeCollapseVertexUVs[iEdge * 3] =     (afVertexUVComponents[iUV0 * 3]     + afVertexUVComponents[iUV1 * 3]) * 0.5f;
        afRetEdgeCollapseVertexUVs[iEdge * 3 + 1] = (afVertexUVComponents[iUV0 * 3 + 1] + afVertexUVComponents[iUV1 * 3 + 1]) * 0.5f;
    }

    // compute the cost of the contraction (transpose(v_optimal) * M * v_optimal)
    afRetEdgeCollapseCosts[iEdge] =
        afEdgeQuadrics[0] * afRetEdgeCollapseVertexPositions[iEdge * 3] * afRetEdgeCollapseVertexPositions[iEdge * 3] +
        2.0f * afEdgeQuadrics[1] * afRetEdgeCollapseVertexPositions[iEdge * 3] * afRetEdgeCollapseVertexPositions[iEdge * 3 + 1] +
        2.0f * afEdgeQuadrics[2] * afRetEdgeCollapseVertexPositions[iEdge * 3] * afRetEdgeCollapseVertexPositions[iEdge * 3 + 2] +
        2.0f * afEdgeQuadrics[3] * afRetEdgeCollapseVertexPositions[iEdge * 3] +

        afEdgeQuadrics[5] * afRetEdgeCollapseVertexPositions[iEdge * 3 + 1] * afRetEdgeCollapseVertexPositions[iEdge * 3 + 1] +
        2.0f * afEdgeQuadrics[6] * afRetEdgeCollapseVertexPositions[iEdge * 3 + 1] * afRetEdgeCollapseVertexPositions[iEdge * 3 + 2] +
        2.0f * afEdgeQuadrics[7] * afRetEdgeCollapseVertexPositions[iEdge * 3 + 1] +

        afEdgeQuadrics[10] * afRetEdgeCollapseVertexPositions[iEdge * 3 + 2] * afRetEdgeCollapseVertexPositions[iEdge * 3 + 2] +
        2.0f * afEdgeQuadrics[11] * afRetEdgeCollapseVertexPositions[iEdge * 3 + 2] +

        afEdgeQuadrics[15];

//printf("edge %d collapse cost: %.4f\n", iEdge, afRetEdgeCollapseCosts[iEdge]);
//if(iEdge == 0)
//{
//    printf("edge %d optimal vertex position (%.4f, %.4f, %.4f)\n",
//        iEdge,
//        afRetEdgeCollapseVertexPositions[iEdge * 3],
//        afRetEdgeCollapseVertexPositions[iEdge * 3 + 1],
//        afRetEdgeCollapseVertexPositions[iEdge * 3 + 2]);
//}
}

/*
**
*/
__device__
float _dot(float fX0, float fY0, float fZ0, float fX1, float fY1, float fZ1)
{
    return fX0 * fX1 + fY0 * fY1 + fZ0 * fZ1;
}

/*
**
*/
__device__
float _length(float fX, float fY, float fZ)
{
    return sqrt(fX * fX + fY * fY + fZ * fZ);
}

/*
**
*/
__device__
void _normalize(
    float* fNormalizedX,
    float* fNormalizedY,
    float* fNormalizedZ,
    float fX, 
    float fY, 
    float fZ)
{
    float fLength = _length(fX, fY, fZ);
    *fNormalizedX = fX / fLength;
    *fNormalizedY = fY / fLength;
    *fNormalizedZ = fZ / fLength;
}

/*
**
*/
__device__
void _cross(
    float* fRetX,
    float* fRetY,
    float* fRetZ,
    float fX0,
    float fY0,
    float fZ0,
    float fX1,
    float fY1,
    float fZ1)
{
    *fRetX = fY0 * fZ1 - fZ0 * fY1;
    *fRetY = fZ0 * fX1 - fX0 * fZ1;
    *fRetZ = fX0 * fY1 - fY0 * fX1;
}

/*
**
*/
__global__
void computeQuadrics(
    float* afRetQuadrics,
    float* afRetAdjacentCount,
    uint32_t* aiTriangleVertexPositionIndices,
    float* afVertexPositionComponents,
    uint32_t iNumVertices,
    uint32_t iNumTriangleIndices)
{
    uint32_t iVertex = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iVertex >= iNumVertices)
    {
        return;
    }

//    uint32_t iVertexPositionComponentIndex = iVertex * 3;
//    uint32_t iQuadricComponentIndex = iVertex * 16;

    float fAdjacentCount = 0.0f;
    float afTotalQuadricMatrix[16] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
    float afAverageVertexNormal[3] = { 0.0f, 0.0f, 0.0f };
    for(uint32_t iTri = 0; iTri < iNumTriangleIndices; iTri += 3)
    {
        uint32_t iV0 = aiTriangleVertexPositionIndices[iTri];
        uint32_t iV1 = aiTriangleVertexPositionIndices[iTri + 1];
        uint32_t iV2 = aiTriangleVertexPositionIndices[iTri + 2];

        float afPlane[4] = { 0.0f, 0.0f, 0.0f, 0.0f };
        if(iV0 == iVertex)
        {
            float fDiffX0 = afVertexPositionComponents[iV1 * 3] -       afVertexPositionComponents[iV0 * 3];
            float fDiffY0 = afVertexPositionComponents[iV1 * 3 + 1] -   afVertexPositionComponents[iV0 * 3 + 1];
            float fDiffZ0 = afVertexPositionComponents[iV1 * 3 + 2] -   afVertexPositionComponents[iV0 * 3 + 2];
            float fNormalizedX0, fNormalizedY0, fNormalizedZ0;
            _normalize(&fNormalizedX0, &fNormalizedY0, &fNormalizedZ0, fDiffX0, fDiffY0, fDiffZ0);
            
            float fDiffX1 = afVertexPositionComponents[iV2 * 3] -       afVertexPositionComponents[iV0 * 3];
            float fDiffY1 = afVertexPositionComponents[iV2 * 3 + 1] -   afVertexPositionComponents[iV0 * 3 + 1];
            float fDiffZ1 = afVertexPositionComponents[iV2 * 3 + 2] -   afVertexPositionComponents[iV0 * 3 + 2];
            float fNormalizedX1, fNormalizedY1, fNormalizedZ1;
            _normalize(&fNormalizedX1, &fNormalizedY1, &fNormalizedZ1, fDiffX1, fDiffY1, fDiffZ1);
            
            _cross(&afPlane[0], &afPlane[1], &afPlane[2], fNormalizedX1, fNormalizedY1, fNormalizedZ1, fNormalizedX0, fNormalizedY0, fNormalizedZ0);
            afPlane[3] = _dot(
                afPlane[0], 
                afPlane[1], 
                afPlane[2], 
                afVertexPositionComponents[iV0 * 3], 
                afVertexPositionComponents[iV0 * 3 + 1], 
                afVertexPositionComponents[iV0 * 3 + 2]) * -1.0f;

            afTotalQuadricMatrix[0] += afPlane[0] * afPlane[0];
            afTotalQuadricMatrix[1] += afPlane[0] * afPlane[1];
            afTotalQuadricMatrix[2] += afPlane[0] * afPlane[2];
            afTotalQuadricMatrix[3] += afPlane[0] * afPlane[3];

            afTotalQuadricMatrix[4] += afPlane[1] * afPlane[0];
            afTotalQuadricMatrix[5] += afPlane[1] * afPlane[1];
            afTotalQuadricMatrix[6] += afPlane[1] * afPlane[2];
            afTotalQuadricMatrix[7] += afPlane[1] * afPlane[3];

            afTotalQuadricMatrix[8] += afPlane[2] * afPlane[0];
            afTotalQuadricMatrix[9] += afPlane[2] * afPlane[1];
            afTotalQuadricMatrix[10] += afPlane[2] * afPlane[2];
            afTotalQuadricMatrix[11] += afPlane[2] * afPlane[3];

            afTotalQuadricMatrix[12] += afPlane[3] * afPlane[0];
            afTotalQuadricMatrix[13] += afPlane[3] * afPlane[1];
            afTotalQuadricMatrix[14] += afPlane[3] * afPlane[2];
            afTotalQuadricMatrix[15] += afPlane[3] * afPlane[3];

            afAverageVertexNormal[0] += afPlane[0];
            afAverageVertexNormal[1] += afPlane[1];
            afAverageVertexNormal[2] += afPlane[2];

            fAdjacentCount += 1.0f;
        }
        else if(iV1 == iVertex)
        {
            float fDiffX0 = afVertexPositionComponents[iV0 * 3] -       afVertexPositionComponents[iV1 * 3];
            float fDiffY0 = afVertexPositionComponents[iV0 * 3 + 1] -   afVertexPositionComponents[iV1 * 3 + 1];
            float fDiffZ0 = afVertexPositionComponents[iV0 * 3 + 2] -   afVertexPositionComponents[iV1 * 3 + 2];
            float fNormalizedX0, fNormalizedY0, fNormalizedZ0;
            _normalize(&fNormalizedX0, &fNormalizedY0, &fNormalizedZ0, fDiffX0, fDiffY0, fDiffZ0);

            float fDiffX1 = afVertexPositionComponents[iV2 * 3] -       afVertexPositionComponents[iV1 * 3];
            float fDiffY1 = afVertexPositionComponents[iV2 * 3 + 1] -   afVertexPositionComponents[iV1 * 3 + 1];
            float fDiffZ1 = afVertexPositionComponents[iV2 * 3 + 2] -   afVertexPositionComponents[iV1 * 3 + 2];
            float fNormalizedX1, fNormalizedY1, fNormalizedZ1;
            _normalize(&fNormalizedX1, &fNormalizedY1, &fNormalizedZ1, fDiffX1, fDiffY1, fDiffZ1);

            _cross(&afPlane[0], &afPlane[1], &afPlane[2], fNormalizedX1, fNormalizedY1, fNormalizedZ1, fNormalizedX0, fNormalizedY0, fNormalizedZ0);
            afPlane[3] = _dot(
                afPlane[0],
                afPlane[1],
                afPlane[2],
                afVertexPositionComponents[iV0 * 3],
                afVertexPositionComponents[iV0 * 3 + 1],
                afVertexPositionComponents[iV0 * 3 + 2]) * -1.0f;

            afTotalQuadricMatrix[0] += afPlane[0] * afPlane[0];
            afTotalQuadricMatrix[1] += afPlane[0] * afPlane[1];
            afTotalQuadricMatrix[2] += afPlane[0] * afPlane[2];
            afTotalQuadricMatrix[3] += afPlane[0] * afPlane[3];

            afTotalQuadricMatrix[4] += afPlane[1] * afPlane[0];
            afTotalQuadricMatrix[5] += afPlane[1] * afPlane[1];
            afTotalQuadricMatrix[6] += afPlane[1] * afPlane[2];
            afTotalQuadricMatrix[7] += afPlane[1] * afPlane[3];

            afTotalQuadricMatrix[8] += afPlane[2] * afPlane[0];
            afTotalQuadricMatrix[9] += afPlane[2] * afPlane[1];
            afTotalQuadricMatrix[10] += afPlane[2] * afPlane[2];
            afTotalQuadricMatrix[11] += afPlane[2] * afPlane[3];

            afTotalQuadricMatrix[12] += afPlane[3] * afPlane[0];
            afTotalQuadricMatrix[13] += afPlane[3] * afPlane[1];
            afTotalQuadricMatrix[14] += afPlane[3] * afPlane[2];
            afTotalQuadricMatrix[15] += afPlane[3] * afPlane[3];

            afAverageVertexNormal[0] += afPlane[0];
            afAverageVertexNormal[1] += afPlane[1];
            afAverageVertexNormal[2] += afPlane[2];

            fAdjacentCount += 1.0f;
        }
        else if(iV2 == iVertex)
        {
            float fDiffX0 = afVertexPositionComponents[iV0 * 3] -       afVertexPositionComponents[iV2 * 3];
            float fDiffY0 = afVertexPositionComponents[iV0 * 3 + 1] -   afVertexPositionComponents[iV2 * 3 + 1];
            float fDiffZ0 = afVertexPositionComponents[iV0 * 3 + 2] -   afVertexPositionComponents[iV2 * 3 + 2];
            float fNormalizedX0, fNormalizedY0, fNormalizedZ0;
            _normalize(&fNormalizedX0, &fNormalizedY0, &fNormalizedZ0, fDiffX0, fDiffY0, fDiffZ0);

            float fDiffX1 = afVertexPositionComponents[iV1 * 3] -       afVertexPositionComponents[iV2 * 3];
            float fDiffY1 = afVertexPositionComponents[iV1 * 3 + 1] -   afVertexPositionComponents[iV2 * 3 + 1];
            float fDiffZ1 = afVertexPositionComponents[iV1 * 3 + 2] -   afVertexPositionComponents[iV2 * 3 + 2];
            float fNormalizedX1, fNormalizedY1, fNormalizedZ1;
            _normalize(&fNormalizedX1, &fNormalizedY1, &fNormalizedZ1, fDiffX1, fDiffY1, fDiffZ1);

            _cross(&afPlane[0], &afPlane[1], &afPlane[2], fNormalizedX1, fNormalizedY1, fNormalizedZ1, fNormalizedX0, fNormalizedY0, fNormalizedZ0);
            afPlane[3] = _dot(
                afPlane[0],
                afPlane[1],
                afPlane[2],
                afVertexPositionComponents[iV0 * 3],
                afVertexPositionComponents[iV0 * 3 + 1],
                afVertexPositionComponents[iV0 * 3 + 2]) * -1.0f;

            afTotalQuadricMatrix[0] += afPlane[0] * afPlane[0];
            afTotalQuadricMatrix[1] += afPlane[0] * afPlane[1];
            afTotalQuadricMatrix[2] += afPlane[0] * afPlane[2];
            afTotalQuadricMatrix[3] += afPlane[0] * afPlane[3];

            afTotalQuadricMatrix[4] += afPlane[1] * afPlane[0];
            afTotalQuadricMatrix[5] += afPlane[1] * afPlane[1];
            afTotalQuadricMatrix[6] += afPlane[1] * afPlane[2];
            afTotalQuadricMatrix[7] += afPlane[1] * afPlane[3];

            afTotalQuadricMatrix[8] += afPlane[2] * afPlane[0];
            afTotalQuadricMatrix[9] += afPlane[2] * afPlane[1];
            afTotalQuadricMatrix[10] += afPlane[2] * afPlane[2];
            afTotalQuadricMatrix[11] += afPlane[2] * afPlane[3];

            afTotalQuadricMatrix[12] += afPlane[3] * afPlane[0];
            afTotalQuadricMatrix[13] += afPlane[3] * afPlane[1];
            afTotalQuadricMatrix[14] += afPlane[3] * afPlane[2];
            afTotalQuadricMatrix[15] += afPlane[3] * afPlane[3];

            afAverageVertexNormal[0] += afPlane[0];
            afAverageVertexNormal[1] += afPlane[1];
            afAverageVertexNormal[2] += afPlane[2];

            fAdjacentCount += 1.0f;
        }

    }   // for tri = 0 to num vertex components

    memcpy(&afRetQuadrics[iVertex * 16], &afTotalQuadricMatrix[0], sizeof(float) * 16);
    afRetAdjacentCount[iVertex] = fAdjacentCount;
}

#define MAX_NUM_PLANES_PER_VERTEX     100

/*
**
*/
__global__
void computeAverageVertexNormals(
    float* afRetVertexPlanes,
    float* afRetAverageVertexNormals,
    float* afRetQuadricMatrices,
    uint32_t* aiRetNumVertexPlanes,
    uint32_t* aiTriangleVertexPositionIndices,
    float* afVertexPositionComponents,
    uint32_t iNumVertices,
    uint32_t iNumTriangleIndices)
{
    uint32_t iVertex = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iVertex >= iNumVertices)
    {
        return;
    }

    uint32_t iVertexComponent = iVertex * 3;

    float afAverageNormal[3] = { 0.0f, 0.0f, 0.0f };
    float afTotalQuadricMatrix[16] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,  0.0f, 0.0f, 0.0f, 0.0f,  0.0f, 0.0f, 0.0f, 0.0f };
//    uint32_t iNumPlanes = 0;
    for(uint32_t iTri = 0; iTri < iNumTriangleIndices; iTri += 3)
    {
        uint32_t iV0 = aiTriangleVertexPositionIndices[iTri];
        uint32_t iV1 = aiTriangleVertexPositionIndices[iTri + 1];
        uint32_t iV2 = aiTriangleVertexPositionIndices[iTri + 2];

        if(iV0 == iVertex || iV1 == iVertex || iV2 == iVertex)
        {
            float fDiffX0 = 0.0f, fDiffY0 = 0.0f, fDiffZ0 = 0.0f;
            float fDiffX1 = 0.0f, fDiffY1 = 0.0f, fDiffZ1 = 0.0f;
            if(iV0 == iVertex)
            {
                fDiffX0 = afVertexPositionComponents[iV1 * 3] - afVertexPositionComponents[iV0 * 3];
                fDiffY0 = afVertexPositionComponents[iV1 * 3 + 1] - afVertexPositionComponents[iV0 * 3 + 1];
                fDiffZ0 = afVertexPositionComponents[iV1 * 3 + 2] - afVertexPositionComponents[iV0 * 3 + 2];

                fDiffX1 = afVertexPositionComponents[iV2 * 3] - afVertexPositionComponents[iV0 * 3];
                fDiffY1 = afVertexPositionComponents[iV2 * 3 + 1] - afVertexPositionComponents[iV0 * 3 + 1];
                fDiffZ1 = afVertexPositionComponents[iV2 * 3 + 2] - afVertexPositionComponents[iV0 * 3 + 2];
            }
            else if(iV1 == iVertex)
            {
                fDiffX0 = afVertexPositionComponents[iV0 * 3] -     afVertexPositionComponents[iV1 * 3];
                fDiffY0 = afVertexPositionComponents[iV0 * 3 + 1] - afVertexPositionComponents[iV1 * 3 + 1];
                fDiffZ0 = afVertexPositionComponents[iV0 * 3 + 2] - afVertexPositionComponents[iV1 * 3 + 2];

                fDiffX1 = afVertexPositionComponents[iV2 * 3] -     afVertexPositionComponents[iV1 * 3];
                fDiffY1 = afVertexPositionComponents[iV2 * 3 + 1] - afVertexPositionComponents[iV1 * 3 + 1];
                fDiffZ1 = afVertexPositionComponents[iV2 * 3 + 2] - afVertexPositionComponents[iV1 * 3 + 2];
            }
            else if(iV2 == iVertex)
            {
                fDiffX0 = afVertexPositionComponents[iV0 * 3] -     afVertexPositionComponents[iV2 * 3];
                fDiffY0 = afVertexPositionComponents[iV0 * 3 + 1] - afVertexPositionComponents[iV2 * 3 + 1];
                fDiffZ0 = afVertexPositionComponents[iV0 * 3 + 2] - afVertexPositionComponents[iV2 * 3 + 2];

                fDiffX1 = afVertexPositionComponents[iV1 * 3] -     afVertexPositionComponents[iV2 * 3];
                fDiffY1 = afVertexPositionComponents[iV1 * 3 + 1] - afVertexPositionComponents[iV2 * 3 + 1];
                fDiffZ1 = afVertexPositionComponents[iV1 * 3 + 2] - afVertexPositionComponents[iV2 * 3 + 2];
            }

            float fNormalizedX0, fNormalizedY0, fNormalizedZ0;
            _normalize(&fNormalizedX0, &fNormalizedY0, &fNormalizedZ0, fDiffX0, fDiffY0, fDiffZ0);
                
            float fNormalizedX1, fNormalizedY1, fNormalizedZ1;
            _normalize(&fNormalizedX1, &fNormalizedY1, &fNormalizedZ1, fDiffX1, fDiffY1, fDiffZ1);

            float afPlane[4] = { 0.0f, 0.0f, 0.0f, 0.0f };
            _cross(&afPlane[0], &afPlane[1], &afPlane[2], fNormalizedX1, fNormalizedY1, fNormalizedZ1, fNormalizedX0, fNormalizedY0, fNormalizedZ0);
            afPlane[3] = _dot(
                afPlane[0],
                afPlane[1],
                afPlane[2],
                afVertexPositionComponents[iV0 * 3],
                afVertexPositionComponents[iV0 * 3 + 1],
                afVertexPositionComponents[iV0 * 3 + 2]) * -1.0f;

            afAverageNormal[0] += afPlane[0];
            afAverageNormal[1] += afPlane[1];
            afAverageNormal[2] += afPlane[2];

            uint32_t iCurrNumPlanes = atomicAdd(&aiRetNumVertexPlanes[iVertex], 1);
            uint32_t iVertexPlaneIndex = iVertex * MAX_NUM_PLANES_PER_VERTEX + iCurrNumPlanes * 4;
            afRetVertexPlanes[iVertexPlaneIndex] = afPlane[0];
            afRetVertexPlanes[iVertexPlaneIndex + 1] = afPlane[1];
            afRetVertexPlanes[iVertexPlaneIndex + 2] = afPlane[2];
            afRetVertexPlanes[iVertexPlaneIndex + 3] = afPlane[3];

            afTotalQuadricMatrix[0] += afPlane[0] * afPlane[0];
            afTotalQuadricMatrix[1] += afPlane[0] * afPlane[1];
            afTotalQuadricMatrix[2] += afPlane[0] * afPlane[2];
            afTotalQuadricMatrix[3] += afPlane[0] * afPlane[3];

            afTotalQuadricMatrix[4] += afPlane[1] * afPlane[0];
            afTotalQuadricMatrix[5] += afPlane[1] * afPlane[1];
            afTotalQuadricMatrix[6] += afPlane[1] * afPlane[2];
            afTotalQuadricMatrix[7] += afPlane[1] * afPlane[3];

            afTotalQuadricMatrix[8] += afPlane[2] * afPlane[0];
            afTotalQuadricMatrix[9] += afPlane[2] * afPlane[1];
            afTotalQuadricMatrix[10] += afPlane[2] * afPlane[2];
            afTotalQuadricMatrix[11] += afPlane[2] * afPlane[3];

            afTotalQuadricMatrix[12] += afPlane[3] * afPlane[0];
            afTotalQuadricMatrix[13] += afPlane[3] * afPlane[1];
            afTotalQuadricMatrix[14] += afPlane[3] * afPlane[2];
            afTotalQuadricMatrix[15] += afPlane[3] * afPlane[3];

        }   // if contains vertex 
    
    }   // for tri = 0 to num triangles

    if(afAverageNormal[0] == 0.0f && afAverageNormal[1] == 0.0f && afAverageNormal[2] == 0.0f)
    {
        afRetAverageVertexNormals[iVertexComponent] = 0.0f;
        afRetAverageVertexNormals[iVertexComponent + 1] = 0.0f;
        afRetAverageVertexNormals[iVertexComponent + 2] = 0.0f;
    }
    else
    {
        _normalize(
            &afRetAverageVertexNormals[iVertexComponent],
            &afRetAverageVertexNormals[iVertexComponent + 1],
            &afRetAverageVertexNormals[iVertexComponent + 2],
            afAverageNormal[0],
            afAverageNormal[1],
            afAverageNormal[2]);
    }

    //if(iVertex == 0 || iVertex == 1)
    //{
    //    printf("vertex %d\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n%.4f\n",
    //        iVertex,
    //        afTotalQuadricMatrix[0], afTotalQuadricMatrix[1], afTotalQuadricMatrix[2], afTotalQuadricMatrix[3],
    //        afTotalQuadricMatrix[4], afTotalQuadricMatrix[5], afTotalQuadricMatrix[6], afTotalQuadricMatrix[7],
    //        afTotalQuadricMatrix[8], afTotalQuadricMatrix[9], afTotalQuadricMatrix[10], afTotalQuadricMatrix[11],
    //        afTotalQuadricMatrix[12], afTotalQuadricMatrix[13], afTotalQuadricMatrix[14], afTotalQuadricMatrix[15]);
    //}

    memcpy(
        &afRetQuadricMatrices[iVertex * 16],
        &afTotalQuadricMatrix[0],
        16 * sizeof(float));

    //printf("%d average normal (%.4f, %.4f, %.4f)\n", 
    //    iVertex, 
    //    afRetAverageVertexNormals[iVertexComponent],
    //    afRetAverageVertexNormals[iVertexComponent + 1],
    //    afRetAverageVertexNormals[iVertexComponent + 2]);
}

/*
**
*/
__global__ 
void computeTotalNormalPlaneAngles(
    float* afRetTotalNormalPlaneAngles,
    float* afVertexNormals,
    float* afVertexPlanes,
    uint32_t* aiNumVertexPlanes,
    uint32_t iNumVertices)
{
    uint32_t iVertex = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iVertex >= iNumVertices)
    {
        return;
    }

    uint32_t iNumVertexPlanes = aiNumVertexPlanes[iVertex];
    float fNormalX = afVertexNormals[iVertex * 3];
    float fNormalY = afVertexNormals[iVertex * 3 + 1];
    float fNormalZ = afVertexNormals[iVertex * 3 + 2];

    float fTotalAngle = 0.0f;
    for(uint32_t iPlane = 0; iPlane < iNumVertexPlanes; iPlane++)
    {
        uint32_t iPlaneIndex = iVertex * MAX_NUM_PLANES_PER_VERTEX + iPlane * 4;
        float fPlaneX = afVertexPlanes[iPlaneIndex];
        float fPlaneY = afVertexPlanes[iPlaneIndex + 1];
        float fPlaneZ = afVertexPlanes[iPlaneIndex + 2];
        fTotalAngle += abs(_dot(fNormalX, fNormalY, fNormalZ, fPlaneX, fPlaneY, fPlaneZ));
    }

    afRetTotalNormalPlaneAngles[iVertex] = fTotalAngle / max(float(iNumVertexPlanes), 0.001f);

//printf("%d total normal plane angles: %.4f\n", iVertex, afRetTotalNormalPlaneAngles[iVertex]);
}

/*
**
*/
__global__
void getMatchingTriangleNormalAndUV(
    uint32_t* aiRetNormalIndices,
    uint32_t* aiRetUVIndices,
    uint32_t* aiTriangleVertexPositionIndices,
    uint32_t* aiTriangleVertexNormalIndices,
    uint32_t* aiTriangleVertexUVIndices,
    uint32_t* aiEdges,
    uint32_t iNumEdges,
    uint32_t iNumTriangleVertexPositionIndices)
{
    uint32_t iEdge = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iEdge >= iNumEdges)
    {
        return;
    }

    uint32_t iEdgeComponent = iEdge * 2;

    uint32_t iEdgePos0 = aiEdges[iEdgeComponent];
    uint32_t iEdgePos1 = aiEdges[iEdgeComponent + 1];

    uint32_t iNorm0 = UINT32_MAX, iNorm1 = UINT32_MAX;
    uint32_t iUV0 = UINT32_MAX;
    uint32_t iUV1 = UINT32_MAX;
    {
        uint32_t iMatchingTri = 0;
        uint32_t aiTriIndices[3] = { UINT32_MAX, UINT32_MAX, UINT32_MAX };
        for(iMatchingTri = 0; iMatchingTri < iNumTriangleVertexPositionIndices; iMatchingTri += 3)
        {
            uint32_t iNumSamePosition = 0;
            aiTriIndices[0] = aiTriIndices[1] = aiTriIndices[2] = UINT32_MAX;
            for(uint32_t i = 0; i < 3; i++)
            {
                if(aiTriangleVertexPositionIndices[iMatchingTri + i] == iEdgePos0 ||
                    aiTriangleVertexPositionIndices[iMatchingTri + i] == iEdgePos1)
                {
                    aiTriIndices[iNumSamePosition] = i;
                    ++iNumSamePosition;
                }
            }

            if(iNumSamePosition >= 2)
            {
                break;
            }
        }

        // didn't find a matching triangle for this edge, continue to the next edge
        if(iMatchingTri >= iNumTriangleVertexPositionIndices)
        {
            aiRetNormalIndices[iEdgeComponent] = UINT32_MAX;
            aiRetNormalIndices[iEdgeComponent + 1] = UINT32_MAX;

            aiRetUVIndices[iEdgeComponent] = UINT32_MAX;
            aiRetUVIndices[iEdgeComponent + 1] = UINT32_MAX;

            return;
        }

        iNorm0 = aiTriangleVertexNormalIndices[iMatchingTri + aiTriIndices[0]];
        iNorm1 = aiTriangleVertexNormalIndices[iMatchingTri + aiTriIndices[1]];

        iUV0 = aiTriangleVertexUVIndices[iMatchingTri + aiTriIndices[0]];
        iUV1 = aiTriangleVertexUVIndices[iMatchingTri + aiTriIndices[1]];

        aiRetNormalIndices[iEdgeComponent] = iNorm0;
        aiRetNormalIndices[iEdgeComponent + 1] = iNorm1;

        aiRetUVIndices[iEdgeComponent] = iUV0;
        aiRetUVIndices[iEdgeComponent + 1] = iUV1;
    }
}

/*
**
*/
__global__
void getShortestVertexDistances(
    float* afRetShortestDistances,
    uint32_t* aiRetShortestVertexPositionIndices,
    float* afVertexPositions0,
    float* afVertexPositions1,
    uint32_t iNumVertexPositions0,
    uint32_t iNumVertexPositions1)
{
    uint32_t iVertex = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iVertex >= iNumVertexPositions0)
    {
        return;
    }

    uint32_t iVertexComponentIndex = iVertex * 3;
    float fX = afVertexPositions0[iVertexComponentIndex];
    float fY = afVertexPositions0[iVertexComponentIndex + 1];
    float fZ = afVertexPositions0[iVertexComponentIndex + 2];

    float fShortestLength = 1.0e+10f;
    uint32_t iNumVertexPositionComponents1 = iNumVertexPositions1 * 3;
    for(uint32_t iPos = 0; iPos < iNumVertexPositionComponents1; iPos += 3)
    {
        float fCheckX = afVertexPositions1[iPos];
        float fCheckY = afVertexPositions1[iPos + 1];
        float fCheckZ = afVertexPositions1[iPos + 2];

        float fLength = _length(fCheckX - fX, fCheckY - fY, fCheckZ - fZ);
        
        if(fLength < fShortestLength)
        {
            afRetShortestDistances[iVertex] = fLength;
            aiRetShortestVertexPositionIndices[iVertex] = iPos / 3;

            fShortestLength = fLength;
        }
    }
}


/*
**
*/
__global__
void buildClusterEdgeAdjacency(
    uint32_t* paaiRetAdjacentEdgeClusters,
    uint32_t* paaiRetNumAdjacentEdgeClusters,
    float* pafTotalClusterVertexPositions,
    uint32_t* paaiVertexPositionIndices,
    uint32_t* paiNumVertexPositionComponents,
    uint32_t* paiNumVertexPositionIndices,
    uint32_t* paiVertexPositionComponentOffsets,
    uint32_t* paiVertexPositionIndexOffsets,
    uint32_t* paiDistanceSortedCluster,
    uint32_t iNumClusters)
{
    uint32_t iCluster = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iCluster >= iNumClusters)
    {
        return;
    }

//printf("start buildClusterEdgeAdjacency processing cluster %d of total %d clusters\n", iCluster, iNumClusters);

    uint32_t iVertexPositionComponentOffset = paiVertexPositionComponentOffsets[iCluster];
    uint32_t iVertexPositionIndexOffset = paiVertexPositionIndexOffsets[iCluster];

    uint32_t iNumTri = paiNumVertexPositionIndices[iCluster];
    for(uint32_t iTri = 0; iTri < iNumTri; iTri += 3)
    {
//        uint32_t iTriComponent = iVertexPositionIndexOffset + iTri * 3;
        //for(uint32_t iCheckCluster = iCluster + 1; iCheckCluster < iNumClusters; iCheckCluster++)

        for(uint32_t iCheckCluster = 0; iCheckCluster < 10; iCheckCluster++)
        {
            uint32_t iCheckClusterID = paiDistanceSortedCluster[iCluster * iNumClusters + iCheckCluster];
            if(iCheckClusterID == iCluster)
            {
                continue;
            }

            uint32_t iCheckVertexPositionComponentOffset = paiVertexPositionComponentOffsets[iCheckClusterID];
            uint32_t iCheckVertexPositionIndexOffset = paiVertexPositionIndexOffsets[iCheckClusterID];

            uint32_t iNumCheckTri = paiNumVertexPositionIndices[iCheckClusterID];
            for(uint32_t iCheckTri = 0; iCheckTri < iNumCheckTri; iCheckTri += 3)
            {
                // check same positions for the triangles
//                uint32_t iCheckTriComponent = iCheckTri * 3;
                //uint32_t aiSamePos[3];
                //uint32_t aiPos[3];
                uint32_t iNumSamePos = 0;
                for(uint32_t i = 0; i < 3; i++)
                {
                    uint32_t iPos = paaiVertexPositionIndices[iVertexPositionIndexOffset + iTri];
                    uint32_t iCheckPos = paaiVertexPositionIndices[iCheckVertexPositionIndexOffset + iCheckTri];

                    float fX = pafTotalClusterVertexPositions[iVertexPositionComponentOffset + (iPos + i) * 3];
                    float fY = pafTotalClusterVertexPositions[iVertexPositionComponentOffset + (iPos + i) * 3 + 1];
                    float fZ = pafTotalClusterVertexPositions[iVertexPositionComponentOffset + (iPos + i) * 3 + 2];
                    
//if(iTri == 3)
//{
//    printf("cluster %d vertex %d (%.4f, %.4f, %.4f) tri: %d local index: %d : total index: %d\n",
//        iCluster,
//        iPos,
//        fX, fY, fZ,
//        iTri,
//        iPos,
//        iVertexPositionComponentOffset + iPos * 3);
//}
                    for(uint32_t j = 0; j < 3; j++)
                    {
                        float fCheckX = pafTotalClusterVertexPositions[iCheckVertexPositionComponentOffset + (iCheckPos + j) * 3];
                        float fCheckY = pafTotalClusterVertexPositions[iCheckVertexPositionComponentOffset + (iCheckPos + j) * 3 + 1];
                        float fCheckZ = pafTotalClusterVertexPositions[iCheckVertexPositionComponentOffset + (iCheckPos + j) * 3 + 2];

                        float fLength = _length(fX - fCheckX, fY - fCheckY, fZ - fCheckZ);
                        if(fLength <= 1.0e-7f)
                        {
                            //aiPos[i] = iPos + i;
                            //aiSamePos[i] = iCheckPos + j;
                            ++iNumSamePos;
                        }
                    }
                }

                // check if 2 or more positions are the same, ie. same edge
                if(iNumSamePos >= 2)
                {
                    uint32_t iNumAdjacentEdgeClusters = atomicAdd(&paaiRetNumAdjacentEdgeClusters[iCluster], 1);
                    uint32_t iNumCheckAdjacentEdgeClusters = atomicAdd(&paaiRetNumAdjacentEdgeClusters[iCheckClusterID], 1);

                    paaiRetAdjacentEdgeClusters[iCluster * iNumClusters + iCheckClusterID] = 1;
                    paaiRetAdjacentEdgeClusters[iCheckClusterID * iNumClusters + iCluster] = 1;

//printf("cluster %d is adjacent to cluster %d vertex id (%d, %d, %d) (%d, %d, %d)\n",
//    iCluster,
//    iCheckCluster,
//    aiPos[0],
//    aiPos[1],
//    aiPos[2],
//    aiSamePos[0],
//    aiSamePos[1],
//    aiSamePos[2]);

                    break;
                }

            }   // for check tri

        }   // for check cluster

    }   // for tri

//printf("end buildClusterEdgeAdjacency processing cluster %d of total %d clusters\n", iCluster, iNumClusters);
    
}

/*
**
*/
__global__
void getClusterBounds(
    float* pafRetMinMaxCenterPositions,
    float* pafTotalClusterVertexPositions,
    uint32_t* paiVertexPositionComponentOffsets,
    uint32_t* paiNumVertexPositionComponents,
    uint32_t iNumClusters)
{
    uint32_t iCluster = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iCluster >= iNumClusters)
    {
        return;
    }

    float fLargestX = -1.0e+10f, fLargestY = -1.0e+10f, fLargestZ = -1.0e+10f;
    float fSmallestX = 1.0e+10f, fSmallestY = 1.0e+10f, fSmallestZ = 1.0e+10f;

    uint32_t iVertexPositionComponentOffset = paiVertexPositionComponentOffsets[iCluster];
    for(uint32_t i = 0; i < paiNumVertexPositionComponents[iCluster]; i += 3)
    {
        float fX = pafTotalClusterVertexPositions[iVertexPositionComponentOffset + i];
        float fY = pafTotalClusterVertexPositions[iVertexPositionComponentOffset + i + 1];
        float fZ = pafTotalClusterVertexPositions[iVertexPositionComponentOffset + i + 2];

        fLargestX = max(fLargestX, fX);
        fLargestY = max(fLargestY, fY);
        fLargestZ = max(fLargestZ, fZ);

        fSmallestX = min(fSmallestX, fX);
        fSmallestY = min(fSmallestY, fY);
        fSmallestZ = min(fSmallestZ, fZ);
    }
    
    uint32_t iIndex = iCluster * 10;
    pafRetMinMaxCenterPositions[iIndex]     = fSmallestX;
    pafRetMinMaxCenterPositions[iIndex + 1] = fSmallestY;
    pafRetMinMaxCenterPositions[iIndex + 2] = fSmallestZ;

    pafRetMinMaxCenterPositions[iIndex + 3] = fLargestX;
    pafRetMinMaxCenterPositions[iIndex + 4] = fLargestY;
    pafRetMinMaxCenterPositions[iIndex + 5] = fLargestZ;

    pafRetMinMaxCenterPositions[iIndex + 6] = (fLargestX + fSmallestX) * 0.5f;
    pafRetMinMaxCenterPositions[iIndex + 7] = (fLargestY + fSmallestY) * 0.5f;
    pafRetMinMaxCenterPositions[iIndex + 8] = (fLargestZ + fSmallestZ) * 0.5f;

    pafRetMinMaxCenterPositions[iIndex + 9] = _length(fLargestX - fSmallestX, fLargestY - fSmallestY, fLargestZ - fSmallestZ) * 0.5f;
}

/*
**
*/
__global__
void getClusterDistances(
    float* pafRetDistances,
    float* pafClusterCenters,
    uint32_t iNumClusters)
{
    uint32_t iCluster = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iCluster >= iNumClusters)
    {
        return;
    }

    float fX = pafClusterCenters[iCluster * 3];
    float fY = pafClusterCenters[iCluster * 3 + 1];
    float fZ = pafClusterCenters[iCluster * 3 + 2];

    for(uint32_t iCheckCluster = 0; iCheckCluster < iNumClusters; iCheckCluster++)
    {
        if(iCheckCluster == iCluster)
        {
            continue;
        }

        float fCheckX = pafClusterCenters[iCheckCluster * 3];
        float fCheckY = pafClusterCenters[iCheckCluster * 3 + 1];
        float fCheckZ = pafClusterCenters[iCheckCluster * 3 + 2];
        float fLength = _length(fX - fCheckX, fY - fCheckY, fZ - fCheckZ);

        uint32_t iIndex = iCluster * iNumClusters + iCheckCluster;
        pafRetDistances[iIndex] = fLength;
    }
}

#include "float3_lib.cuh"

/*
**
*/
__global__
void projectVertices(
    float* afRetProjectedPositions,
    float* afTriangleVertexPositions0,
    float* afTriangleVertexPositions1,
    float* afIntersectInfo,
    uint32_t iNumVertices0,
    uint32_t iNumVertices1)
{
    uint32_t iNumTriangles = iNumVertices0 / 3;
    uint32_t iTriangle = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iTriangle >= iNumTriangles)
    {
        return;
    }

    float3 pos0 = make_float3(
        afTriangleVertexPositions0[iTriangle * 9], 
        afTriangleVertexPositions0[iTriangle * 9 + 1], 
        afTriangleVertexPositions0[iTriangle * 9 + 2]);
    float3 pos1 = make_float3(
        afTriangleVertexPositions0[iTriangle * 9 + 3], 
        afTriangleVertexPositions0[iTriangle * 9 + 4], 
        afTriangleVertexPositions0[iTriangle * 9 + 5]);
    float3 pos2 = make_float3(
        afTriangleVertexPositions0[iTriangle * 9 + 6], 
        afTriangleVertexPositions0[iTriangle * 9 + 7], 
        afTriangleVertexPositions0[iTriangle * 9 + 8]);
    
    float3 diff0 = normalize(pos2 - pos0);
    float3 diff1 = normalize(pos1 - pos0);

    float3 faceNormal = cross(diff0, diff1);

    uint32_t iNumCheckTriangles = iNumVertices1 / 3;
    for(uint32_t i = 0; i < 3; i++)
    {
        float3 ret = make_float3(FLT_MAX, FLT_MAX, FLT_MAX);

        uint32_t iTriangleVertexIndex = iTriangle * 9 + i * 3;
        float3 pos = make_float3(
            afTriangleVertexPositions0[iTriangleVertexIndex],
            afTriangleVertexPositions0[iTriangleVertexIndex + 1],
            afTriangleVertexPositions0[iTriangleVertexIndex + 2]);
        
        float fT = FLT_MAX;
        float fIntersectTriangle = FLT_MAX;
        for(uint32_t iCheckTriangle = 0; iCheckTriangle < iNumCheckTriangles; iCheckTriangle++)
        {
            float3 checkPos0 = make_float3(
                afTriangleVertexPositions1[iCheckTriangle * 9],
                afTriangleVertexPositions1[iCheckTriangle * 9 + 1],
                afTriangleVertexPositions1[iCheckTriangle * 9 + 2]);
            float3 checkPos1 = make_float3(
                afTriangleVertexPositions1[iCheckTriangle * 9 + 3],
                afTriangleVertexPositions1[iCheckTriangle * 9 + 4],
                afTriangleVertexPositions1[iCheckTriangle * 9 + 5]);
            float3 checkPos2 = make_float3(
                afTriangleVertexPositions1[iCheckTriangle * 9 + 6],
                afTriangleVertexPositions1[iCheckTriangle * 9 + 7],
                afTriangleVertexPositions1[iCheckTriangle * 9 + 8]);

            float3 diff0 = normalize(checkPos2 - checkPos0);
            float3 diff1 = normalize(checkPos1 - checkPos0);

            // normal of the plane
            float3 checkNormal = cross(diff0, diff1);
            float fPlaneD = dot(checkNormal, checkPos0) * -1.0f;

            float3 pt1 = pos + faceNormal * 100.0f;
            fT = rayPlaneIntersection(
                pos,
                pt1,
                checkNormal,
                fPlaneD);

            // account for forward and backward direction
            if(fT >= -1.0f && fT <= 1.0f)
            {
                float3 intersectionPt = pos + (pt1 - pos) * fT;
                float3 barycentricPt = barycentric(intersectionPt, checkPos0, checkPos1, checkPos2);
                bool bProjected =
                    (barycentricPt.x >= -0.01f && barycentricPt.x <= 1.01f &&
                        barycentricPt.y >= -0.01f && barycentricPt.y <= 1.01f &&
                        barycentricPt.z >= -0.01f && barycentricPt.z <= 1.01f);
                if(bProjected)
                {
                    ret = checkPos0 * barycentricPt.x + checkPos1 * barycentricPt.y + checkPos2 * barycentricPt.z;
                    fIntersectTriangle = float(iCheckTriangle);
                }
            }

            if(ret.x != FLT_MAX)
            {
                break;
            }
        }

        if(ret.x == FLT_MAX)
        {
            ret = pos + faceNormal * 10.0f;
            afRetProjectedPositions[iTriangleVertexIndex] = ret.x;
            afRetProjectedPositions[iTriangleVertexIndex+1] = ret.y;
            afRetProjectedPositions[iTriangleVertexIndex+2] = ret.z;
        }
        else
        {
            afRetProjectedPositions[iTriangleVertexIndex] = ret.x;
            afRetProjectedPositions[iTriangleVertexIndex + 1] = ret.y;
            afRetProjectedPositions[iTriangleVertexIndex + 2] = ret.z;
        }

        afIntersectInfo[iTriangle * 3 * 2 + i * 2] = fT;
        afIntersectInfo[iTriangle * 3 * 2 + i * 2 + 1] = fIntersectTriangle;

    }   // for i = 0 to 3
}

/*
**
*/
__global__
void buildClusterEdgeAdjacency2(
    uint32_t* paaiRetAdjacentEdgeClusters,
    uint32_t* paaiRetNumAdjacentEdgeClusters,
    float* pafTotalClusterVertexPositions,
    uint32_t* paaiVertexPositionIndices,
    uint32_t* paiNumVertexPositionComponents,
    uint32_t* paiNumVertexPositionIndices,
    uint32_t* paiVertexPositionComponentOffsets,
    uint32_t* paiVertexPositionIndexOffsets,
    uint32_t* paiDistanceSortedCluster,
    uint32_t iNumClusters)
{
    uint32_t iCluster = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iCluster >= iNumClusters)
    {
        return;
    }

    uint32_t iVertexPositionComponentOffset = paiVertexPositionComponentOffsets[iCluster];
    uint32_t iVertexPositionIndexOffset = paiVertexPositionIndexOffsets[iCluster];

    uint32_t iNumTri = paiNumVertexPositionIndices[iCluster];
    for(uint32_t iTri = 0; iTri < iNumTri; iTri += 3)
    {
        for(uint32_t iCheckCluster = 0; iCheckCluster < 10; iCheckCluster++)
        {
            uint32_t iCheckClusterID = paiDistanceSortedCluster[iCluster * iNumClusters + iCheckCluster];
            if(iCheckClusterID == iCluster)
            {
                continue;
            }

            uint32_t iCheckVertexPositionComponentOffset = paiVertexPositionComponentOffsets[iCheckClusterID];
            uint32_t iCheckVertexPositionIndexOffset = paiVertexPositionIndexOffsets[iCheckClusterID];

            uint32_t iNumSamePos = 0;
            uint32_t iNumCheckTri = paiNumVertexPositionIndices[iCheckClusterID];
            for(uint32_t iCheckTri = 0; iCheckTri < iNumCheckTri; iCheckTri += 3)
            {
                // check same positions for the triangles
                for(uint32_t i = 0; i < 3; i++)
                {
                    uint32_t iPos = paaiVertexPositionIndices[iVertexPositionIndexOffset + iTri];
                    uint32_t iCheckPos = paaiVertexPositionIndices[iCheckVertexPositionIndexOffset + iCheckTri];

                    float fX = pafTotalClusterVertexPositions[iVertexPositionComponentOffset + (iPos + i) * 3];
                    float fY = pafTotalClusterVertexPositions[iVertexPositionComponentOffset + (iPos + i) * 3 + 1];
                    float fZ = pafTotalClusterVertexPositions[iVertexPositionComponentOffset + (iPos + i) * 3 + 2];

                    bool bSame = false;
                    for(uint32_t j = 0; j < 3; j++)
                    {
                        float fCheckX = pafTotalClusterVertexPositions[iCheckVertexPositionComponentOffset + (iCheckPos + j) * 3];
                        float fCheckY = pafTotalClusterVertexPositions[iCheckVertexPositionComponentOffset + (iCheckPos + j) * 3 + 1];
                        float fCheckZ = pafTotalClusterVertexPositions[iCheckVertexPositionComponentOffset + (iCheckPos + j) * 3 + 2];

                        float fLength = _length(fX - fCheckX, fY - fCheckY, fZ - fCheckZ);
                        if(fLength <= 1.0e-8f)
                        {
                            ++iNumSamePos;
                            bSame = true;
                            break;
                        }
                    }

                    if(bSame)
                    {
                        break;
                    }
                }

            }   // for check tri

            if(iNumSamePos > 0)
            {
                paaiRetAdjacentEdgeClusters[iCluster * iNumClusters + iCheckClusterID] = iNumSamePos;
            }

        }   // for check cluster

    }   // for tri
}

/*
**
*/
__global__
void checkClusterGroupAdjacency2(
    uint32_t* aiAdjacentClusterGroupVertexIndices,
    uint32_t* aiNumAdjacentClusterGroupVertices,
    float3 const* afTotalClusterGroupVertexPositionComponents,
    uint32_t const* aiNumVertexPositions,
    uint32_t const* aiClusterGroupVertexPositionArrayByteOffsets,
    uint32_t iNumTotalClusterGroups)
{
    uint32_t iClusterGroup = blockIdx.x * WORKGROUP_SIZE + threadIdx.x;
    if(iClusterGroup >= iNumTotalClusterGroups)
    {
        return;
    }
    
    uint32_t iVertexPositionByteOffset = aiClusterGroupVertexPositionArrayByteOffsets[iClusterGroup];
    float3 const* aClusterVertexPositions = (float3 const*)((char*)afTotalClusterGroupVertexPositionComponents + iVertexPositionByteOffset);
    uint32_t iNumVertexPositions = aiNumVertexPositions[iClusterGroup];
    
    uint32_t iNumAdjacentClusterGroupVertices = 0;
    for(uint32_t iVertex = 0; iVertex < iNumVertexPositions; iVertex++)
    {
        float3 const& position = aClusterVertexPositions[iVertex];
        bool bHasAdjacentVertex = false;
        for(uint32_t iCheckClusterGroup = 0; iCheckClusterGroup < iNumTotalClusterGroups; iCheckClusterGroup++)
        {
            if(iCheckClusterGroup == iClusterGroup)
            {
                continue;
            }

            uint32_t iCheckVertexPositionByteOffset = aiClusterGroupVertexPositionArrayByteOffsets[iCheckClusterGroup];
            float3 const* aCheckClusterVertexPositions = (float3 const*)((char*)afTotalClusterGroupVertexPositionComponents + iCheckVertexPositionByteOffset);
            uint32_t iNumCheckVertexPositions = aiNumVertexPositions[iCheckClusterGroup];

            for(uint32_t iCheckVertex = 0; iCheckVertex < iNumCheckVertexPositions; iCheckVertex++)
            {
                float3 const& checkPosition = aCheckClusterVertexPositions[iCheckVertex];
                float3 diff = position - checkPosition;

                if(lengthSquared(diff) <= 1.0e-10f)
                {
if(iNumAdjacentClusterGroupVertices >= MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP)
{
    printf("wtf\n");
}
                    
                    uint32_t iIndex = iClusterGroup * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP + iNumAdjacentClusterGroupVertices;
                    aiAdjacentClusterGroupVertexIndices[iIndex] = iVertex;

                    ++iNumAdjacentClusterGroupVertices;
                    bHasAdjacentVertex = true;

                    break;
                }

            }   // for check vertex component = 0 to num vertex components

            if(bHasAdjacentVertex)
            {
                break;
            }

        }   // for check cluster group = 0 to num cluster groups 
    
    }   // for vertex = 0 to num vertices in cluster group

    aiNumAdjacentClusterGroupVertices[iClusterGroup] = iNumAdjacentClusterGroupVertices;
}

#undef uint32_t
#undef int32_t

#include "test.h"
#include "LogPrint.h"

#include <chrono>
#include <assert.h>

/*
**
*/
void checkClusterGroupBoundaryVerticesCUDA(
    std::vector<std::vector<uint32_t>>& aaiClusterGroupBoundaryVertices, 
    std::vector<std::vector<vec3>> const& aaClusterGroupVertexPositions,
    std::vector<std::vector<uint32_t>> const& aaiClusterGroupTrianglePositionIndices)
{
DEBUG_PRINTF("*** checkClusterGroupBoundaryVerticesCUDA ***\n");
auto start = std::chrono::high_resolution_clock::now();

    PrintOptions printOptions;
    printOptions.mbDisplayTime = false;
    setPrintOptions(printOptions);

    // prepare data to be passed into device, getting data offsets and the number of vertex position for clusters and number of triangle indices
    uint32_t iCurrVertexPositionDataOffset = 0;
    uint32_t iCurrTriangleIndexDataOffset = 0;
    uint32_t iNumClusterGroups = static_cast<uint32_t>(aaClusterGroupVertexPositions.size());
    uint32_t iNumTotalTriangleIndices = 0;
    std::vector<uint32_t> aiVertexPositionDataOffsets(iNumClusterGroups);
    std::vector<uint32_t> aiTriangleIndexOffsets(iNumClusterGroups);
    std::vector<uint32_t> aiNumVertexPositions(iNumClusterGroups);
    std::vector<uint32_t> aiNumTriangleIndices(iNumClusterGroups);
    for(uint32_t iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        aiVertexPositionDataOffsets[iClusterGroup] = iCurrVertexPositionDataOffset;
        aiTriangleIndexOffsets[iClusterGroup] = iCurrTriangleIndexDataOffset;

        aiNumVertexPositions[iClusterGroup] = static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size());
        aiNumTriangleIndices[iClusterGroup] = static_cast<uint32_t>(aaiClusterGroupTrianglePositionIndices[iClusterGroup].size());

        //DEBUG_PRINTF("position offset cluster group %d : %d\n", iClusterGroup, iCurrVertexPositionDataOffset);
        //DEBUG_PRINTF("triangle index offset cluster group %d : %d\n", iClusterGroup, iCurrTriangleIndexDataOffset);

        iCurrVertexPositionDataOffset += static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size() * 3);
        iCurrTriangleIndexDataOffset += static_cast<uint32_t>(aaiClusterGroupTrianglePositionIndices[iClusterGroup].size());

        iNumTotalTriangleIndices += aiNumTriangleIndices[iClusterGroup];
    }
    
    //DEBUG_PRINTF("total position offset : %d\n", iCurrVertexPositionDataOffset);
    //DEBUG_PRINTF("total triangle index offset : %d\n", iCurrVertexPositionDataOffset);
    //DEBUG_PRINTF("total num triangle indices: %d\n", iNumTotalTriangleIndices);

    // allocate device memory
    float* pafTotalClusterGroupVertexPositions;
    hipMalloc(&pafTotalClusterGroupVertexPositions, iCurrVertexPositionDataOffset * sizeof(float));
    
    unsigned int* paiTotalClusterGroupTriangleIndices;
    hipMalloc(&paiTotalClusterGroupTriangleIndices, iCurrTriangleIndexDataOffset * sizeof(int));

    unsigned int* paiNumClusterGroupVertexPositions;
    hipMalloc(&paiNumClusterGroupVertexPositions, iNumClusterGroups * sizeof(int));

    unsigned int* paiNumClusterGroupTriangleIndices;
    hipMalloc(&paiNumClusterGroupTriangleIndices, iNumClusterGroups * sizeof(int));

    unsigned int* paiClusterGroupVertexPositionDataOffsets;
    hipMalloc(&paiClusterGroupVertexPositionDataOffsets, iNumClusterGroups * sizeof(int));

    unsigned int* paiClusterGroupTriangleIndexDataOffsets;
    hipMalloc(&paiClusterGroupTriangleIndexDataOffsets, iNumClusterGroups * sizeof(int));

    unsigned int* aiRetNumClusterGroupBoundaryVertices;
    hipMalloc(&aiRetNumClusterGroupBoundaryVertices, iNumClusterGroups * sizeof(int));

    unsigned int* aiRetClusterGroupBoundaryVertexIndices;
    hipMalloc(&aiRetClusterGroupBoundaryVertexIndices, iNumClusterGroups * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP * sizeof(int));

    // copy over data (vertex positions, triangle indices) for all the cluster groups
    iCurrVertexPositionDataOffset = 0;
    iCurrTriangleIndexDataOffset = 0;
    for(uint32_t iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        hipMemcpy(
            pafTotalClusterGroupVertexPositions + iCurrVertexPositionDataOffset,
            aaClusterGroupVertexPositions[iClusterGroup].data(),
            aaClusterGroupVertexPositions[iClusterGroup].size() * sizeof(float3),
            hipMemcpyHostToDevice);

        hipMemcpy(
            paiTotalClusterGroupTriangleIndices + iCurrTriangleIndexDataOffset,
            aaiClusterGroupTrianglePositionIndices[iClusterGroup].data(),
            aaiClusterGroupTrianglePositionIndices[iClusterGroup].size() * sizeof(uint32_t),
            hipMemcpyHostToDevice);

        iCurrVertexPositionDataOffset += static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size() * 3);
        iCurrTriangleIndexDataOffset += static_cast<uint32_t>(aaiClusterGroupTrianglePositionIndices[iClusterGroup].size());
    }

    // vertex positions
    hipMemcpy(
        paiNumClusterGroupVertexPositions,
        aiNumVertexPositions.data(),
        aiNumVertexPositions.size() * sizeof(int),
        hipMemcpyHostToDevice);

    // triangle indices
    hipMemcpy(
        paiNumClusterGroupTriangleIndices,
        aiNumTriangleIndices.data(),
        aiNumTriangleIndices.size() * sizeof(int),
        hipMemcpyHostToDevice);

    // vertex position offset for all the cluster groups
    hipMemcpy(
        paiClusterGroupVertexPositionDataOffsets,
        aiVertexPositionDataOffsets.data(),
        aiVertexPositionDataOffsets.size() * sizeof(int),
        hipMemcpyHostToDevice);

    // triangle index offsets for all the cluster groups
    hipMemcpy(
        paiClusterGroupTriangleIndexDataOffsets,
        aiTriangleIndexOffsets.data(),
        aiTriangleIndexOffsets.size() * sizeof(int),
        hipMemcpyHostToDevice);

    // run on device
    //uint32_t iNumBlocks = static_cast<uint32_t>(std::max(1, int32_t(iNumClusterGroups) / 256));
    //checkClusterGroupBoundaryVertices<<<iNumBlocks, 256>>>(
    //    aiRetClusterGroupBoundaryVertexIndices,
    //    aiRetNumClusterGroupBoundaryVertices,
    //    iNumClusterGroups,
    //    pafTotalClusterGroupVertexPositions,
    //    paiTotalClusterGroupTriangleIndices,
    //    paiNumClusterGroupVertexPositions,
    //    paiNumClusterGroupTriangleIndices,
    //    paiClusterGroupVertexPositionDataOffsets,
    //    paiClusterGroupTriangleIndexDataOffsets);

    // initialize
    std::vector<uint32_t> aiDefaultValues(iNumClusterGroups);
    hipMemcpy(
        aiRetNumClusterGroupBoundaryVertices,
        aiDefaultValues.data(),
        iNumClusterGroups * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(float(iNumTotalTriangleIndices / 3) / float(WORKGROUP_SIZE)));
    checkClusterGroupBoundaryVertices2<<<iNumBlocks, WORKGROUP_SIZE>>>(
        aiRetClusterGroupBoundaryVertexIndices,
        aiRetNumClusterGroupBoundaryVertices,
        iNumClusterGroups,
        pafTotalClusterGroupVertexPositions,
        paiTotalClusterGroupTriangleIndices,
        paiNumClusterGroupVertexPositions,
        paiNumClusterGroupTriangleIndices,
        paiClusterGroupVertexPositionDataOffsets,
        paiClusterGroupTriangleIndexDataOffsets,
        iNumTotalTriangleIndices);

    // copy output
    std::vector<uint32_t> aiRetClusterGroupBoundaryVertexIndicesCPU(iNumClusterGroups * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP);
    std::vector<uint32_t> aiRetNumClusterGroupBoundaryVerticesCPU(iNumClusterGroups);
    hipMemcpy(
        aiRetClusterGroupBoundaryVertexIndicesCPU.data(), 
        aiRetClusterGroupBoundaryVertexIndices, 
        iNumClusterGroups * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP * sizeof(uint32_t),
        hipMemcpyDeviceToHost);
    hipMemcpy(
        aiRetNumClusterGroupBoundaryVerticesCPU.data(), 
        aiRetNumClusterGroupBoundaryVertices, 
        iNumClusterGroups * sizeof(uint32_t),
        hipMemcpyDeviceToHost);

    hipFree(pafTotalClusterGroupVertexPositions);
    hipFree(paiTotalClusterGroupTriangleIndices);
    hipFree(paiNumClusterGroupVertexPositions);
    hipFree(paiNumClusterGroupTriangleIndices);
    hipFree(paiClusterGroupVertexPositionDataOffsets);
    hipFree(paiClusterGroupTriangleIndexDataOffsets);
    hipFree(aiRetNumClusterGroupBoundaryVertices);
    hipFree(aiRetClusterGroupBoundaryVertexIndices);

    // output cluster group boundary vertex indices
    aaiClusterGroupBoundaryVertices.resize(iNumClusterGroups);
    for(uint32_t iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        uint32_t iDataOffset = iClusterGroup * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP;
        for(uint32_t i = 0; i < aiRetNumClusterGroupBoundaryVerticesCPU[iClusterGroup]; i++)
        {
            uint32_t iPos = aiRetClusterGroupBoundaryVertexIndicesCPU[iDataOffset + i];
            auto iter = std::find(
                aaiClusterGroupBoundaryVertices[iClusterGroup].begin(),
                aaiClusterGroupBoundaryVertices[iClusterGroup].end(),
                iPos);
            if(iter == aaiClusterGroupBoundaryVertices[iClusterGroup].end())
            {
                aaiClusterGroupBoundaryVertices[iClusterGroup].push_back(iPos);

                //vec3 const& pos = aaClusterGroupVertexPositions[iClusterGroup][iPos];
                //DEBUG_PRINTF("\tdraw_sphere([%.4f, %.4f, %.4f], 0.01, 255, 0, 0)\n",
                //    pos.x, pos.y, pos.z);
            }
        }

        std::sort(aaiClusterGroupBoundaryVertices[iClusterGroup].begin(), aaiClusterGroupBoundaryVertices[iClusterGroup].end());
    }

    printOptions.mbDisplayTime = true;
    setPrintOptions(printOptions);

auto end = std::chrono::high_resolution_clock::now();
uint64_t iSeconds = std::chrono::duration_cast<std::chrono::seconds>(end - start).count();
DEBUG_PRINTF("*** took %lld seconds for checkClusterGroupBoundaryVerticesCUDA to complete ***\n", iSeconds);

}

/*
**
*/
void buildClusterAdjacencyCUDA(
    std::vector<std::vector<uint32_t>>& aaiNumAdjacentVertices,
    std::vector<std::vector<vec3>> const& aaVertexPositions,
    bool bOnlyEdgeAdjacent)
{
DEBUG_PRINTF("*** start buildClusterAdjacencyCUDA ***\n");
auto start = std::chrono::high_resolution_clock::now();

    uint32_t iCurrVertexPositionDataOffset = 0;
    uint32_t iNumClusters = static_cast<uint32_t>(aaVertexPositions.size());
    std::vector<uint32_t> aiNumVertexPositionComponents(iNumClusters);
    std::vector<uint32_t> aiVertexPositionComponentArrayOffsets(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        aiNumVertexPositionComponents[iCluster] = static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);
        aiVertexPositionComponentArrayOffsets[iCluster] = iCurrVertexPositionDataOffset;
        
        //DEBUG_PRINTF("cluster %d num vertices: %d num vertex components: %d data offset: %d\n",
        //    iCluster,
        //    aaVertexPositions[iCluster].size(),
        //    aiNumVertexPositionComponents[iCluster],
        //    iCurrVertexPositionDataOffset);
        
        iCurrVertexPositionDataOffset += static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);
    }

    // allocate device memory
    float* pafTotalClusterVertexPositions;
    uint32_t iArrayIndexOffset = 0;
    hipMalloc(&pafTotalClusterVertexPositions, iCurrVertexPositionDataOffset * sizeof(float));
    
    uint32_t* paiNumAdjacentClusterVertices;
    hipMalloc(&paiNumAdjacentClusterVertices, iNumClusters * iNumClusters * sizeof(int));
    
    // initialize
    //std::vector<uint32_t> aiDefaultValues(iNumClusters * iNumClusters);
    //memset(aiDefaultValues.data(), 0, iNumClusters * iNumClusters * sizeof(int));
    //hipMemcpy(
    //    paiNumAdjacentClusterVertices,
    //    aiDefaultValues.data(),
    //    iNumClusters * iNumClusters * sizeof(int),
    //    hipMemcpyHostToDevice);

    hipMemset(
        paiNumAdjacentClusterVertices,
        0,
        iNumClusters * iNumClusters * sizeof(int));

    uint32_t* paiNumVertexPositionComponents;
    hipMalloc(&paiNumVertexPositionComponents, iNumClusters * sizeof(int));
    
    uint32_t* paiNumVertexPositionComponentOffsets;
    hipMalloc(&paiNumVertexPositionComponentOffsets, iNumClusters * sizeof(int));

    // copy to device memory
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        hipMemcpy(
            pafTotalClusterVertexPositions + iArrayIndexOffset,
            aaVertexPositions[iCluster].data(),
            aaVertexPositions[iCluster].size() * sizeof(float) * 3,
            hipMemcpyHostToDevice);
        iArrayIndexOffset += static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);
    }

    hipMemcpy(
        paiNumVertexPositionComponents,
        aiNumVertexPositionComponents.data(),
        aiNumVertexPositionComponents.size() * sizeof(int),
        hipMemcpyHostToDevice);

    hipMemcpy(
        paiNumVertexPositionComponentOffsets,
        aiVertexPositionComponentArrayOffsets.data(),
        aiVertexPositionComponentArrayOffsets.size() * sizeof(int),
        hipMemcpyHostToDevice);


    //void checkClusterAdjacency(
//    float* afTotalClusterVertexPositionComponents,
//    uint32_t * aiNumAdjacentVertices,
//    uint32_t * aiNumVertexPositionComponents,
//    uint32_t * aiClusterVertexPositionComponentOffsets,
//    uint32_t iNumTotalClusters)

    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumClusters) / float(WORKGROUP_SIZE)));
    checkClusterAdjacency<<<iNumBlocks, WORKGROUP_SIZE>>>(
        paiNumAdjacentClusterVertices,
        pafTotalClusterVertexPositions,
        paiNumVertexPositionComponents,
        paiNumVertexPositionComponentOffsets,
        iNumClusters,
        bOnlyEdgeAdjacent);

    std::vector<uint32_t> aiNumAdjacentClusterVerticesCPU(iNumClusters * iNumClusters);
    hipMemcpy(
        aiNumAdjacentClusterVerticesCPU.data(),
        paiNumAdjacentClusterVertices,
        iNumClusters * iNumClusters * sizeof(int),
        hipMemcpyDeviceToHost);

    aaiNumAdjacentVertices.resize(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        aaiNumAdjacentVertices[iCluster].resize(iNumClusters);
        for(uint32_t iCheckCluster = 0; iCheckCluster < iNumClusters; iCheckCluster++)
        {
            uint32_t iIndex = iCluster * iNumClusters + iCheckCluster;
            aaiNumAdjacentVertices[iCluster][iCheckCluster] = aiNumAdjacentClusterVerticesCPU[iIndex];
        }
    }

    hipFree(paiNumVertexPositionComponentOffsets);
    hipFree(paiNumVertexPositionComponents);
    hipFree(pafTotalClusterVertexPositions);
    hipFree(paiNumAdjacentClusterVertices);

auto end = std::chrono::high_resolution_clock::now();
uint64_t iSeconds = std::chrono::duration_cast<std::chrono::seconds>(end - start).count();
DEBUG_PRINTF("*** took %lld seconds for buildClusterAdjacencyCUDA to finish ***\n",
    iSeconds);

}

/*
**
*/
void getClusterGroupBoundaryVerticesCUDA2(
    std::vector<std::vector<uint32_t>>& aaiClusterGroupBoundaryVertices,
    std::vector<std::vector<vec3>> const& aaClusterGroupVertexPositions)
{
    DEBUG_PRINTF("*** start getClusterGroupBoundaryVerticesCUDA2 ***\n");
    auto start = std::chrono::high_resolution_clock::now();

    uint32_t iNumClusterGroups = static_cast<uint32_t>(aaClusterGroupVertexPositions.size());

    uint32_t iCurrVertexPositionDataOffset = 0;
    std::vector<uint32_t> aiNumVertexPositions(iNumClusterGroups);

    for(uint32_t iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        aiNumVertexPositions[iClusterGroup] = static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size());
        iCurrVertexPositionDataOffset += static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size() * sizeof(float3));
    }

    // copy to device memory
    uint32_t* paiRetClusterGroupAdjacentVertexIndices = nullptr;
    hipMalloc(
        &paiRetClusterGroupAdjacentVertexIndices,
        iNumClusterGroups * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP * sizeof(uint32_t));
    
    uint32_t* paiRetNumAdjacentClusterGroupVertices = nullptr;
    hipMalloc(
        &paiRetNumAdjacentClusterGroupVertices,
        iNumClusterGroups * sizeof(uint32_t));
    
    uint32_t iTotalVertexPositionSize = 0;
    for(uint32_t iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        iTotalVertexPositionSize += static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size() * sizeof(float3));
    }

    float3* paClusterGroupVertexPositions = nullptr;
    hipMalloc(
        &paClusterGroupVertexPositions,
        iTotalVertexPositionSize);
    uint32_t iByteOffset = 0;
    for(uint32_t iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        hipMemcpy(
            (char *)paClusterGroupVertexPositions + iByteOffset,
            aaClusterGroupVertexPositions[iClusterGroup].data(),
            aaClusterGroupVertexPositions[iClusterGroup].size() * sizeof(float3),
            hipMemcpyHostToDevice);
        iByteOffset += static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size() * sizeof(float3));
    }

    uint32_t* paiNumClusterGroupVertexPositions = nullptr;
    hipMalloc(
        &paiNumClusterGroupVertexPositions,
        iNumClusterGroups * sizeof(uint32_t));
    iByteOffset = 0;
    for(uint32_t i = 0; i < iNumClusterGroups; i++)
    {
        uint32_t iNumVertexPositions = static_cast<uint32_t>(aaClusterGroupVertexPositions[i].size());
        hipMemcpy(
            paiNumClusterGroupVertexPositions + i,
            &iNumVertexPositions,
            sizeof(uint32_t),
            hipMemcpyHostToDevice);
    }

    uint32_t* paiClusterGroupVertexPositionArrayByteOffsets = nullptr;
    hipMalloc(
        &paiClusterGroupVertexPositionArrayByteOffsets,
        iNumClusterGroups * sizeof(uint32_t));
    iByteOffset = 0;
    for(uint32_t i = 0; i < iNumClusterGroups; i++)
    {
        hipMemcpy(
            paiClusterGroupVertexPositionArrayByteOffsets + i,
            &iByteOffset,
            sizeof(uint32_t),
            hipMemcpyHostToDevice);
        uint32_t iDataSize = static_cast<uint32_t>(aaClusterGroupVertexPositions[i].size() * sizeof(float3));
        iByteOffset += iDataSize;
    }

    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumClusterGroups) / float(WORKGROUP_SIZE)));
    checkClusterGroupAdjacency2<<<iNumBlocks, WORKGROUP_SIZE>>>(
        paiRetClusterGroupAdjacentVertexIndices,
        paiRetNumAdjacentClusterGroupVertices,
        paClusterGroupVertexPositions,
        paiNumClusterGroupVertexPositions,
        paiClusterGroupVertexPositionArrayByteOffsets,
        iNumClusterGroups);

    std::vector<uint32_t> aiTotalClusterGroupAdjacentVerticesCPU(MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP * iNumClusterGroups);
    hipMemcpy(
        aiTotalClusterGroupAdjacentVerticesCPU.data(),
        paiRetClusterGroupAdjacentVertexIndices,
        iNumClusterGroups * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP * sizeof(uint32_t),
        hipMemcpyDeviceToHost);

    std::vector<uint32_t> aiNumAdjacentVertices(iNumClusterGroups);
    hipMemcpy(
        aiNumAdjacentVertices.data(),
        paiRetNumAdjacentClusterGroupVertices,
        iNumClusterGroups * sizeof(int),
        hipMemcpyDeviceToHost);
    
    iByteOffset = 0;
    aaiClusterGroupBoundaryVertices.resize(iNumClusterGroups);
    for(uint32_t iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        uint32_t iNumBoundaryVertices = aiNumAdjacentVertices[iClusterGroup];
        aaiClusterGroupBoundaryVertices[iClusterGroup].resize(iNumBoundaryVertices);
        memcpy(
            aaiClusterGroupBoundaryVertices[iClusterGroup].data(),
            (char*)aiTotalClusterGroupAdjacentVerticesCPU.data() + iByteOffset,
            iNumBoundaryVertices * sizeof(uint32_t));

        iByteOffset += static_cast<uint32_t>(MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP * sizeof(uint32_t));
    }

    hipFree(paiRetClusterGroupAdjacentVertexIndices);
    hipFree(paiRetNumAdjacentClusterGroupVertices);
    hipFree(paClusterGroupVertexPositions);
    hipFree(paiNumClusterGroupVertexPositions);
}

/*
**
*/
void getClusterGroupBoundaryVerticesCUDA(
    std::vector<std::vector<uint32_t>>& aaiClusterGroupBoundaryVertices,
    std::vector<std::vector<vec3>> const& aaClusterGroupVertexPositions)
{
DEBUG_PRINTF("*** start getClusterGroupBoundaryVerticesCUDA ***\n");
auto start = std::chrono::high_resolution_clock::now();

    uint32_t iNumClusterGroups = static_cast<uint32_t>(aaClusterGroupVertexPositions.size());

    uint32_t iCurrVertexPositionDataOffset = 0;
    std::vector<uint32_t> aiNumVertexPositionComponents(iNumClusterGroups);
    std::vector<uint32_t> aiVertexPositionComponentArrayOffsets(iNumClusterGroups);
    
    uint32_t iNumTotalVertices = 0;
    for(uint32_t iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        aiNumVertexPositionComponents[iClusterGroup] = static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size() * 3);
        aiVertexPositionComponentArrayOffsets[iClusterGroup] = iCurrVertexPositionDataOffset;

        //DEBUG_PRINTF("cluster group %d num vertices: %d num vertex components: %d data offset: %d\n",
        //    iClusterGroup,
        //    aaClusterGroupVertexPositions[iClusterGroup].size(),
        //    aiNumVertexPositionComponents[iClusterGroup],
        //    iCurrVertexPositionDataOffset);

        iCurrVertexPositionDataOffset += static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size() * 3);

        iNumTotalVertices += static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size());
    }

    //DEBUG_PRINTF("num total vertices: %d\n", iNumTotalVertices);

    // allocate device memory
    float* pafTotalClusterVertexPositions;
    uint32_t iArrayIndexOffset = 0;
    hipMalloc(&pafTotalClusterVertexPositions, iCurrVertexPositionDataOffset * sizeof(float));
    
    uint32_t* paiNumAdjacentClusterVertices;
    hipMalloc(&paiNumAdjacentClusterVertices, iNumClusterGroups * iNumClusterGroups * sizeof(int));

    uint32_t* paiNumVertexPositionComponents;
    hipMalloc(&paiNumVertexPositionComponents, iNumClusterGroups * sizeof(int));

    uint32_t* paiNumVertexPositionComponentOffsets;
    hipMalloc(&paiNumVertexPositionComponentOffsets, iNumClusterGroups * sizeof(int));

    uint32_t* paiClusterGroupAdjacentVertexIndices;
    hipMalloc(&paiClusterGroupAdjacentVertexIndices, iNumClusterGroups * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP * sizeof(int));

    // copy to device memory
    for(uint32_t iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        hipMemcpy(
            pafTotalClusterVertexPositions + iArrayIndexOffset,
            aaClusterGroupVertexPositions[iClusterGroup].data(),
            aaClusterGroupVertexPositions[iClusterGroup].size() * sizeof(float) * 3,
            hipMemcpyHostToDevice);
        iArrayIndexOffset += static_cast<uint32_t>(aaClusterGroupVertexPositions[iClusterGroup].size() * 3);
    }

    hipMemcpy(
        paiNumVertexPositionComponents,
        aiNumVertexPositionComponents.data(),
        aiNumVertexPositionComponents.size() * sizeof(int),
        hipMemcpyHostToDevice);

    hipMemcpy(
        paiNumVertexPositionComponentOffsets,
        aiVertexPositionComponentArrayOffsets.data(),
        aiVertexPositionComponentArrayOffsets.size() * sizeof(int),
        hipMemcpyHostToDevice);


    //void checkClusterGroupAdjacency(
    //    uint32_t * aiAdjacentClusterGroupVertexIndices,
    //    float* afTotalClusterGroupVertexPositionComponents,
    //    uint32_t * aiNumAdjacentClusterGroupVertices,
    //    uint32_t * aiNumVertexPositionComponents,
    //    uint32_t * aiClusterGroupVertexPositionComponentOffsets,
    //    uint32_t iNumTotalVertexIndices,
    //    uint32_t iNumTotalClusterGroups)

    std::vector<uint32_t> aiDefaultValues(iNumClusterGroups * iNumClusterGroups);
    memset(aiDefaultValues.data(), 0, aiDefaultValues.size() * sizeof(int));
    hipMemcpy(
        paiNumAdjacentClusterVertices, 
        aiDefaultValues.data(), 
        iNumClusterGroups * iNumClusterGroups * sizeof(int), 
        hipMemcpyHostToDevice);

    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumTotalVertices) / float(WORKGROUP_SIZE)));
    checkClusterGroupAdjacency <<<iNumBlocks, WORKGROUP_SIZE>>>(
        paiClusterGroupAdjacentVertexIndices,
        paiNumAdjacentClusterVertices,
        pafTotalClusterVertexPositions,
        paiNumVertexPositionComponents,
        paiNumVertexPositionComponentOffsets,
        iCurrVertexPositionDataOffset,
        iNumClusterGroups);

    std::vector<uint32_t> aiTotalClusterGroupAdjacentVerticesCPU(iNumClusterGroups * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP);
    hipMemcpy(
        aiTotalClusterGroupAdjacentVerticesCPU.data(),
        paiClusterGroupAdjacentVertexIndices,
        iNumClusterGroups * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP * sizeof(int),
        hipMemcpyDeviceToHost);

    std::vector<uint32_t> aiNumAdjacentVertices(iNumClusterGroups);
    hipMemcpy(
        aiNumAdjacentVertices.data(),
        paiNumAdjacentClusterVertices,
        iNumClusterGroups * sizeof(int),
        hipMemcpyDeviceToHost);

    aaiClusterGroupBoundaryVertices.resize(iNumClusterGroups);
    for(uint32_t iClusterGroup = 0; iClusterGroup < iNumClusterGroups; iClusterGroup++)
    {
        uint32_t iNumAdjacentVertices = aiNumAdjacentVertices[iClusterGroup];
        aaiClusterGroupBoundaryVertices[iClusterGroup].resize(iNumAdjacentVertices);
        memcpy(
            aaiClusterGroupBoundaryVertices[iClusterGroup].data(),
            aiTotalClusterGroupAdjacentVerticesCPU.data() + iClusterGroup * MAX_BOUNDARY_VERTICES_PER_CLUSTER_GROUP,
            sizeof(int) * iNumAdjacentVertices);
        for(auto const& iVertex : aaiClusterGroupBoundaryVertices[iClusterGroup])
        {
            assert(iVertex < aaClusterGroupVertexPositions[iClusterGroup].size());
        }
    }

    hipFree(paiNumAdjacentClusterVertices);
    hipFree(pafTotalClusterVertexPositions);
    hipFree(paiNumVertexPositionComponents);
    hipFree(paiNumVertexPositionComponentOffsets);
    hipFree(paiClusterGroupAdjacentVertexIndices);

auto end = std::chrono::high_resolution_clock::now();
uint64_t iSeconds = std::chrono::duration_cast<std::chrono::seconds>(end - start).count();
DEBUG_PRINTF("*** took %lld seconds for getClusterGroupBoundaryVerticesCUDA to finish ***\n",
    iSeconds);

}

/*
**
*/
void computeEdgeCollapseInfoCUDA(
    std::vector<float>& afCollapseCosts,
    std::vector<vec3>& aOptimalVertexPositions,
    std::vector<vec3>& aOptimalVertexNormals,
    std::vector<vec2>& aOptimalVertexUVs,
    std::vector<std::pair<uint32_t, uint32_t>>& aEdges,
    std::vector<vec3> const& aClusterGroupVertexPositions,
    std::vector<vec3> const& aClusterGroupVertexNormals,
    std::vector<vec2> const& aClusterGroupVertexUVs,
    std::vector<std::pair<uint32_t, uint32_t>> const& aiValidClusterGroupEdgePairs,
    std::vector<uint32_t> const& aiClusterGroupNonBoundaryVertices,
    std::vector<uint32_t> const& aiClusterGroupTrianglePositionIndices,
    std::vector<uint32_t> const& aiClusterGroupTriangleNormalIndices,
    std::vector<uint32_t> const& aiClusterGroupTriangleUVIndices,
    std::vector<std::pair<uint32_t, uint32_t>> const& aBoundaryVertices)
{
//DEBUG_PRINTF("*** start computeEdgeCollapseInfoCUDA ***\n");
//auto start = std::chrono::high_resolution_clock::now();
    
    uint32_t iNumEdges = static_cast<uint32_t>(aiValidClusterGroupEdgePairs.size());
    uint32_t iNumClusteGroupTrianglePositionIndices = static_cast<uint32_t>(aiClusterGroupTrianglePositionIndices.size());
    uint32_t iNumClusterGroupNonBoundaryVertices = static_cast<uint32_t>(aiClusterGroupNonBoundaryVertices.size());
    uint32_t iNumVertices = static_cast<uint32_t>(aClusterGroupVertexPositions.size());
    uint32_t iNumTriangleIndices = static_cast<uint32_t>(aiClusterGroupTrianglePositionIndices.size());

    float* afVertexPositionComponents;
    hipMalloc(
        &afVertexPositionComponents,
        aClusterGroupVertexPositions.size() * 3 * sizeof(float));
    hipMemcpy(
        afVertexPositionComponents,
        aClusterGroupVertexPositions.data(),
        aClusterGroupVertexPositions.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    float* afVertexNormalComponents;
    hipMalloc(
        &afVertexNormalComponents,
        aClusterGroupVertexNormals.size() * 3 * sizeof(float));
    hipMemcpy(
        afVertexNormalComponents,
        aClusterGroupVertexNormals.data(),
        aClusterGroupVertexNormals.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    float* afVertexUVComponents;
    hipMalloc(
        &afVertexUVComponents,
        aClusterGroupVertexUVs.size() * 2 * sizeof(float));
    hipMemcpy(
        afVertexUVComponents,
        aClusterGroupVertexUVs.data(),
        aClusterGroupVertexUVs.size() * 2 * sizeof(float),
        hipMemcpyHostToDevice);

    uint32_t* aiClusterGroupEdgePairs;
    hipMalloc(
        &aiClusterGroupEdgePairs,
        aiValidClusterGroupEdgePairs.size() * 2 * sizeof(int));
    std::vector<uint32_t> aiEdgePairs;
    for(uint32_t i = 0; i < static_cast<uint32_t>(aiValidClusterGroupEdgePairs.size()); i++)
    {
        aiEdgePairs.push_back(aiValidClusterGroupEdgePairs[i].first);
        aiEdgePairs.push_back(aiValidClusterGroupEdgePairs[i].second);
    }
    hipMemcpy(
        aiClusterGroupEdgePairs,
        aiEdgePairs.data(),
        aiEdgePairs.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* aiClusterGroupNonBoundaryVertexIndices;
    hipMalloc(
        &aiClusterGroupNonBoundaryVertexIndices,
        aiClusterGroupNonBoundaryVertices.size() * sizeof(int));
    hipMemcpy(
        aiClusterGroupNonBoundaryVertexIndices,
        aiClusterGroupNonBoundaryVertices.data(),
        aiClusterGroupNonBoundaryVertices.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* aiClusterGroupTrianglePositionIndicesGPU;
    hipMalloc(
        &aiClusterGroupTrianglePositionIndicesGPU,
        aiClusterGroupTrianglePositionIndices.size() * sizeof(int));
    hipMemcpy(
        aiClusterGroupTrianglePositionIndicesGPU,
        aiClusterGroupTrianglePositionIndices.data(),
        aiClusterGroupTrianglePositionIndices.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* aiClusterGroupTriangleNormalIndicesGPU;
    hipMalloc(
        &aiClusterGroupTriangleNormalIndicesGPU,
        aiClusterGroupTriangleNormalIndices.size() * sizeof(int));
    hipMemcpy(
        aiClusterGroupTriangleNormalIndicesGPU,
        aiClusterGroupTriangleNormalIndices.data(),
        aiClusterGroupTriangleNormalIndices.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* aiClusterGroupTriangleUVIndicesGPU;
    hipMalloc(
        &aiClusterGroupTriangleUVIndicesGPU,
        aiClusterGroupTriangleUVIndices.size() * sizeof(int));
    hipMemcpy(
        aiClusterGroupTriangleUVIndicesGPU,
        aiClusterGroupTriangleUVIndices.data(),
        aiClusterGroupTriangleUVIndices.size() * sizeof(int),
        hipMemcpyHostToDevice);

    float* afEdgeCollapseCosts;
    hipMalloc(
        &afEdgeCollapseCosts,
        iNumEdges * sizeof(float));

    uint32_t* aiEdgeCollapseVertexIndices0;
    hipMalloc(
        &aiEdgeCollapseVertexIndices0,
        iNumEdges * sizeof(int));

    uint32_t* aiEdgeCollapseVertexIndices1;
    hipMalloc(
        &aiEdgeCollapseVertexIndices1,
        iNumEdges * sizeof(int));

    float* afEdgeCollapseVertexPositions;
    hipMalloc(
        &afEdgeCollapseVertexPositions,
        iNumEdges * 3 * sizeof(float));

    float* afEdgeCollapseVertexNormals;
    hipMalloc(
        &afEdgeCollapseVertexNormals,
        iNumEdges * 3 * sizeof(float));

    float* afEdgeCollapseVertexUVs;
    hipMalloc(
        &afEdgeCollapseVertexUVs,
        iNumEdges * 3 * sizeof(float));

    float* afQuadrics;
    hipMalloc(
        &afQuadrics,
        aClusterGroupVertexPositions.size() * 16 * sizeof(float));

    float* afTotalNormalPlaneAngles;
    hipMalloc(
        &afTotalNormalPlaneAngles,
        aClusterGroupVertexPositions.size() * sizeof(float));

    uint32_t* aiEdgeNormalIndices;
    hipMalloc(
        &aiEdgeNormalIndices,
        aiEdgePairs.size() * sizeof(int));
    
    uint32_t* aiEdgeUVIndices;
    hipMalloc(
        &aiEdgeUVIndices,
        aiEdgePairs.size() * sizeof(int));

    // build normal and uv map to position for edges
    std::vector<uint32_t> aiEdgeNormalIndicesCPU(iNumEdges * 2);
    std::vector<uint32_t> aiEdgeUVIndicesCPU(iNumEdges * 2);
    {
        //void getMatchingTriangleNormalAndUV(
        //    uint32_t * aiRetNormalIndices,
        //    uint32_t * aiRetUVIndices,
        //    uint32_t * aiTriangleVertexPositionIndices,
        //    uint32_t * aiTriangleVertexNormalIndices,
        //    uint32_t * aiTriangleVertexUVIndices,
        //    uint32_t * aiEdges,
        //    uint32_t iNumEdges,
        //    uint32_t iNumTriangleVertexPositionIndices)

        uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumEdges) / float(WORKGROUP_SIZE)));
        getMatchingTriangleNormalAndUV<<<iNumBlocks, WORKGROUP_SIZE>>>(
            aiEdgeNormalIndices,
            aiEdgeUVIndices,
            aiClusterGroupTrianglePositionIndicesGPU,
            aiClusterGroupTriangleNormalIndicesGPU,
            aiClusterGroupTriangleUVIndicesGPU,
            aiClusterGroupEdgePairs,
            iNumEdges,
            iNumClusteGroupTrianglePositionIndices);

        hipDeviceSynchronize();

        hipMemcpy(
            aiEdgeNormalIndicesCPU.data(),
            aiEdgeNormalIndices,
            sizeof(int) * iNumEdges * 2,
            hipMemcpyDeviceToHost);

        hipMemcpy(
            aiEdgeUVIndicesCPU.data(),
            aiEdgeUVIndices,
            sizeof(int)* iNumEdges * 2,
            hipMemcpyDeviceToHost);
    }

    float* afVertexPlanes;
    float* afAverageVertexNormals;
    uint32_t* aiNumVertexPlanes;
    float* afVertexNormalPlaneAngles;
    {
        hipMalloc(
            &afAverageVertexNormals,
            iNumVertices * sizeof(float) * 3);
        hipMemset(afAverageVertexNormals, 0, iNumVertices * sizeof(float) * 3);

        hipMalloc(
            &afVertexPlanes,
            iNumVertices* MAX_NUM_PLANES_PER_VERTEX * 4 * sizeof(float));
        hipMemset(afVertexPlanes, 0, iNumVertices* MAX_NUM_PLANES_PER_VERTEX * 4 * sizeof(float));

        hipMalloc(
            &aiNumVertexPlanes,
            iNumVertices * sizeof(int));
        hipMemset(aiNumVertexPlanes, 0, iNumVertices * sizeof(int));

        uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumVertices) / float(WORKGROUP_SIZE)));
        iNumBlocks = std::max(iNumBlocks, 1u);
        computeAverageVertexNormals<<<iNumBlocks, WORKGROUP_SIZE>>>(
            afVertexPlanes,
            afAverageVertexNormals,
            afQuadrics,
            aiNumVertexPlanes,
            aiClusterGroupTrianglePositionIndicesGPU,
            afVertexPositionComponents,
            iNumVertices,
            iNumTriangleIndices);

        hipDeviceSynchronize();

        hipMalloc(
            &afVertexNormalPlaneAngles, 
            iNumVertices * sizeof(float));
        
        computeTotalNormalPlaneAngles<<<iNumBlocks, WORKGROUP_SIZE>>>(
            afVertexNormalPlaneAngles,
            afAverageVertexNormals,
            afVertexPlanes,
            aiNumVertexPlanes,
            iNumVertices);

        hipDeviceSynchronize();

        std::vector<float> afVertexPlanesCPU(iNumVertices * MAX_NUM_PLANES_PER_VERTEX * 4);
        std::vector<float> afAverageVertexNormalCPU(iNumVertices * 3);
        std::vector<uint32_t> aiNumVertexPlanesCPU(iNumVertices);
        std::vector<float> afVertexNormalPlaneAnglesCPU(iNumVertices);
        std::vector<float> afQuadricsCPU(iNumVertices * 16);
        hipMemcpy(
            afVertexPlanesCPU.data(),
            afVertexPlanes,
            afVertexPlanesCPU.size() * sizeof(float),
            hipMemcpyDeviceToHost);

        hipMemcpy(
            afAverageVertexNormalCPU.data(),
            afAverageVertexNormals,
            afAverageVertexNormalCPU.size() * sizeof(float),
            hipMemcpyDeviceToHost);

        hipMemcpy(
            aiNumVertexPlanesCPU.data(),
            aiNumVertexPlanes,
            aiNumVertexPlanesCPU.size() * sizeof(int),
            hipMemcpyDeviceToHost);

        hipMemcpy(
            afVertexNormalPlaneAnglesCPU.data(),
            afVertexNormalPlaneAngles,
            afVertexNormalPlaneAnglesCPU.size() * sizeof(float),
            hipMemcpyDeviceToHost);

        hipMemcpy(
            afQuadricsCPU.data(),
            afQuadrics,
            afQuadricsCPU.size() * sizeof(float),
            hipMemcpyDeviceToHost);
    }


    float* afVertexAdjacentTriangleCounts;
    hipMalloc(
        &afVertexAdjacentTriangleCounts,
        iNumVertices * sizeof(float));
    hipMemset(
        afVertexAdjacentTriangleCounts,
        0,
        iNumVertices * sizeof(float));

#if 0
    {
        uint32_t iNumVertices = static_cast<uint32_t>(aClusterGroupVertexPositions.size());
        uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumVertices) / float(WORKGROUP_SIZE)));
        computeQuadrics<<<iNumBlocks, WORKGROUP_SIZE>>>(
            afQuadrics,
            afVertexAdjacentTriangleCounts,
            aiClusterGroupTrianglePositionIndicesGPU,
            afVertexPositionComponents,
            iNumVertices,
            iNumTriangleIndices);

        hipDeviceSynchronize();

        std::vector<float> afVertexAdjacentTriangleCountsCPU(iNumVertices);
        hipMemcpy(
            afVertexAdjacentTriangleCountsCPU.data(),
            afVertexAdjacentTriangleCounts,
            iNumVertices * sizeof(float),
            hipMemcpyDeviceToHost);

        std::vector<float> afQuadricsCPU(iNumVertices * 16);
        hipMemcpy(
            afQuadricsCPU.data(),
            afQuadrics,
            iNumVertices * 16 * sizeof(float),
            hipMemcpyDeviceToHost);

        std::vector<float> afAverageVertexNormalsCPU(iNumVertices * 3);
        hipMemcpy(
            afAverageVertexNormalsCPU.data(),
            afAverageVertexNormals,
            iNumVertices * 3 * sizeof(float),
            hipMemcpyDeviceToHost);

        int iDebug = 1;
    }
#endif // #if 0

    //void computeEdgeCollapseInfo(
        //    float* afRetEdgeCollapseCosts,
        //    uint32_t* aiRetEdgeCollapseVertexIndices0,
        //    uint32_t* aiRetEdgeCollapseVertexIndices1,
        //    float* afRetEdgeCollapseVertexPositions,
        //    float* afRetEdgeCollapseVertexNormals,
        //    float* afRetEdgeCollapseVertexUVs,
        //    uint32_t* aiClusterGroupNonBoundaryVertexIndices,
        //    float* afVertexPositionComponents,
        //    float* afVertexNormalComponents,
        //    float* afVertexUVComponents,
        //    float* afQuadrics,
        //    float* afTotalNormalPlaneAngles,
        //    uint32_t* aiClusterGroupEdgePairs,
        //    uint32_t* aiClusterGroupTrianglePositionIndicesGPU,
        //    uint32_t* aiClusterGroupTriangleNormalIndicesGPU,
        //    uint32_t* aiClusterGroupTriangleUVIndicesGPU,
        //    uint32_t* aiNormalIndexToEdgeMap,
        //    uint32_t* aiUVIndexToEdgeMap,
        //    uint32_t iNumClusterGroupTrianglePositionIndices,
        //    uint32_t iNumClusterGroupNonBoundaryVertices,
        //    uint32_t iNumEdges)


    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(aiEdgePairs.size() / 2) / float(WORKGROUP_SIZE)));
    iNumBlocks = std::max(iNumBlocks, 1u);
    computeEdgeCollapseInfo<<<iNumBlocks, WORKGROUP_SIZE>>>(
        afEdgeCollapseCosts,
        aiEdgeCollapseVertexIndices0,
        aiEdgeCollapseVertexIndices1,
        afEdgeCollapseVertexPositions,
        afEdgeCollapseVertexNormals,
        afEdgeCollapseVertexUVs,
        aiClusterGroupNonBoundaryVertexIndices,
        afVertexPositionComponents,
        afVertexNormalComponents,
        afVertexUVComponents,
        afQuadrics,
        afVertexNormalPlaneAngles,
        aiClusterGroupEdgePairs,
        aiClusterGroupTrianglePositionIndicesGPU,
        aiClusterGroupTriangleNormalIndicesGPU,
        aiClusterGroupTriangleUVIndicesGPU,
        aiEdgeNormalIndices,
        aiEdgeUVIndices,
        iNumClusteGroupTrianglePositionIndices,
        iNumClusterGroupNonBoundaryVertices,
        iNumEdges);

    hipDeviceSynchronize();

    afCollapseCosts.resize(iNumEdges);
    aOptimalVertexPositions.resize(iNumEdges);
    aOptimalVertexNormals.resize(iNumEdges);
    aOptimalVertexUVs.resize(iNumEdges);
    aEdges.resize(iNumEdges);

    hipMemcpy(
        afCollapseCosts.data(),
        afEdgeCollapseCosts,
        afCollapseCosts.size() * sizeof(float),
        hipMemcpyDeviceToHost);

    hipMemcpy(
        aOptimalVertexPositions.data(),
        afEdgeCollapseVertexPositions,
        aOptimalVertexPositions.size() * 3 * sizeof(float),
        hipMemcpyDeviceToHost);

    hipMemcpy(
        aOptimalVertexNormals.data(),
        afEdgeCollapseVertexNormals,
        aOptimalVertexNormals.size() * 3 * sizeof(float),
        hipMemcpyDeviceToHost);

    hipMemcpy(
        aOptimalVertexUVs.data(),
        afEdgeCollapseVertexUVs,
        aOptimalVertexUVs.size() * 2 * sizeof(float),
        hipMemcpyDeviceToHost);

    std::vector<uint32_t> aiEdgeIndex0(iNumEdges);
    std::vector<uint32_t> aiEdgeIndex1(iNumEdges);
    hipMemcpy(
        aiEdgeIndex0.data(),
        aiEdgeCollapseVertexIndices0,
        aiEdgeIndex0.size() * sizeof(int),
        hipMemcpyDeviceToHost);
    hipMemcpy(
        aiEdgeIndex1.data(),
        aiEdgeCollapseVertexIndices1,
        aiEdgeIndex1.size() * sizeof(int),
        hipMemcpyDeviceToHost);

    aEdges.resize(iNumEdges);
    for(uint32_t i = 0; i < iNumEdges; i++)
    {
        aEdges[i] = std::make_pair(aiEdgeIndex0[i], aiEdgeIndex1[i]);
    }

    hipFree(afVertexPositionComponents);
    hipFree(afVertexNormalComponents);
    hipFree(afVertexUVComponents);
    hipFree(aiClusterGroupEdgePairs);
    hipFree(aiClusterGroupNonBoundaryVertexIndices);
    hipFree(aiClusterGroupTrianglePositionIndicesGPU);
    hipFree(aiClusterGroupTriangleNormalIndicesGPU);
    hipFree(aiClusterGroupTriangleUVIndicesGPU);
    hipFree(afEdgeCollapseCosts);
    hipFree(aiEdgeCollapseVertexIndices0);
    hipFree(aiEdgeCollapseVertexIndices1);
    hipFree(afEdgeCollapseVertexPositions);
    hipFree(afEdgeCollapseVertexNormals);
    hipFree(afEdgeCollapseVertexUVs);
    hipFree(afQuadrics);
    hipFree(afTotalNormalPlaneAngles);

    hipFree(aiEdgeNormalIndices);
    hipFree(aiEdgeUVIndices);
    hipFree(afVertexAdjacentTriangleCounts);
    hipFree(afAverageVertexNormals);
    hipFree(afVertexPlanes);
    hipFree(aiNumVertexPlanes);
    hipFree(afVertexNormalPlaneAngles);

//auto end = std::chrono::high_resolution_clock::now();
//uint64_t iSeconds = std::chrono::duration_cast<std::chrono::seconds>(end - start).count();
//DEBUG_PRINTF("*** took %d seconds for computeEdgeCollapseInfoCUDA to finish ***\n",
//    iSeconds);

}

/*
**
*/
void getProjectVertexDistancesCUDA(
    std::vector<vec3>& aProjectedPositions,
    std::vector<vec3> const& aTriangleVertexPositions0,
    std::vector<vec3> const& aTriangleVertexPositions1)
{
    uint32_t iNumVertices0 = static_cast<uint32_t>(aTriangleVertexPositions0.size());
    uint32_t iNumVertices1 = static_cast<uint32_t>(aTriangleVertexPositions1.size());

    assert(iNumVertices0 % 3 == 0);
    assert(iNumVertices1 % 3 == 0);

    float* afRetProjectedPositions;
    hipMalloc(&afRetProjectedPositions, iNumVertices0 * sizeof(float) * 3);
    hipMemset(afRetProjectedPositions, 0, iNumVertices0 * sizeof(float) * 3);

    float* afTriangleVertexPositions0;
    hipMalloc(&afTriangleVertexPositions0, iNumVertices0 * 3 * sizeof(float));
    hipMemcpy(
        afTriangleVertexPositions0,
        aTriangleVertexPositions0.data(),
        aTriangleVertexPositions0.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    float* afTriangleVertexPositions1;
    hipMalloc(&afTriangleVertexPositions1, iNumVertices1 * 3 * sizeof(float));
    hipMemcpy(
        afTriangleVertexPositions1,
        aTriangleVertexPositions1.data(),
        aTriangleVertexPositions1.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    float* afIntersectInfo;
    hipMalloc(&afIntersectInfo, iNumVertices0 * 2 * sizeof(float));
    hipMemset(&afIntersectInfo, 0, iNumVertices0 * 2 * sizeof(float));

    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumVertices0) / float(WORKGROUP_SIZE)));
    iNumBlocks = std::max(iNumBlocks, 1u);
    projectVertices<<<iNumBlocks, WORKGROUP_SIZE>>>(
        afRetProjectedPositions,
        afTriangleVertexPositions0,
        afTriangleVertexPositions1,
        afIntersectInfo,
        iNumVertices0,
        iNumVertices1);

    aProjectedPositions.resize(iNumVertices0);
    hipMemcpy(
        aProjectedPositions.data(),
        afRetProjectedPositions,
        iNumVertices0 * sizeof(float) * 3,
        hipMemcpyKind::hipMemcpyDeviceToHost);

    struct IntersectInfo
    {
        float mfT;
        float mfTriangle;
    };

    std::vector<IntersectInfo> afIntersectT(iNumVertices0);
    hipMemcpy(
        afIntersectT.data(),
        afIntersectInfo,
        iNumVertices0 * 2 * sizeof(float),
        hipMemcpyKind::hipMemcpyDeviceToHost);

    hipFree(afRetProjectedPositions);
    hipFree(afTriangleVertexPositions1);
    hipFree(afTriangleVertexPositions0);
}

/*
**
*/
void getShortestVertexDistancesCUDA(
    std::vector<float>& afClosestDistances,
    std::vector<uint32_t>& aiClosestVertexPositionIndices,
    std::vector<vec3> const& aVertexPositions0,
    std::vector<vec3> const& aVertexPositions1)
{
    uint32_t iNumVertices0 = static_cast<uint32_t>(aVertexPositions0.size());
    uint32_t iNumVertices1 = static_cast<uint32_t>(aVertexPositions1.size());

    //float* afRetShortestDistances,
    //    uint32_t* aiRetShortestVertexPositionIndices,
    //    float* afVertexPositions0,
    //    float* afVertexPositions1,
    //    uint32_t iNumVertexPositions0,
    //    uint32_t iNumVertexPositions1

    float* afRetShortestDistances;
    hipMalloc(&afRetShortestDistances, iNumVertices0 * sizeof(float));
    hipMemset(afRetShortestDistances, 0, iNumVertices0 * sizeof(float));

    uint32_t* aiRetShortestVertexPositionIndices;
    hipMalloc(&aiRetShortestVertexPositionIndices, iNumVertices0 * sizeof(uint32_t));
    hipMemset(aiRetShortestVertexPositionIndices, 0, iNumVertices0 * sizeof(uint32_t));

    float* afVertexPositions0;
    hipMalloc(&afVertexPositions0, iNumVertices0 * 3 * sizeof(float));
    hipMemcpy(
        afVertexPositions0,
        aVertexPositions0.data(),
        aVertexPositions0.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    float* afVertexPositions1;
    hipMalloc(&afVertexPositions1, iNumVertices1 * 3 * sizeof(float));
    hipMemcpy(
        afVertexPositions1,
        aVertexPositions1.data(),
        aVertexPositions1.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumVertices0) / float(WORKGROUP_SIZE)));
    iNumBlocks = std::max(iNumBlocks, 1u);
    getShortestVertexDistances<<<iNumBlocks, WORKGROUP_SIZE>>>(
        afRetShortestDistances,
        aiRetShortestVertexPositionIndices,
        afVertexPositions0,
        afVertexPositions1,
        iNumVertices0,
        iNumVertices1);

    afClosestDistances.resize(aVertexPositions0.size());
    hipMemcpy(
        afClosestDistances.data(),
        afRetShortestDistances,
        aVertexPositions0.size() * sizeof(float),
        hipMemcpyDeviceToHost);

    aiClosestVertexPositionIndices.resize(aVertexPositions0.size());
    hipMemcpy(
        aiClosestVertexPositionIndices.data(),
        aiRetShortestVertexPositionIndices,
        aiClosestVertexPositionIndices.size() * sizeof(int),
        hipMemcpyDeviceToHost);

    hipFree(afRetShortestDistances);
    hipFree(aiRetShortestVertexPositionIndices);
    hipFree(afVertexPositions0);
    hipFree(afVertexPositions1);
}

#define MAX_NUM_ADJACENT_EDGES 50
#define MAX_NUM_CLUSTER_POSITION_INDICES 1000

/*
**
*/
void getSortedEdgeAdjacentClustersCUDA(
    std::vector<std::vector<uint32_t>>& aaiSortedAdjacentEdgeClusters,
    std::vector<std::vector<vec3>> const& aaVertexPositions,
    std::vector<std::vector<uint32_t>> const& aaiVertexPositionIndices)
{
    DEBUG_PRINTF("*** start getSortedEdgeAdjacentClustersCUDA ***\n");
    auto start = std::chrono::high_resolution_clock::now();

    uint32_t iCurrVertexPositionIndexDataOffset = 0;
    uint32_t iCurrVertexPositionDataOffset = 0;
    uint32_t iNumClusters = static_cast<uint32_t>(aaVertexPositions.size());
    std::vector<uint32_t> aiNumVertexPositionComponents(iNumClusters);
    std::vector<uint32_t> aiVertexPositionComponentArrayOffsets(iNumClusters);
    std::vector<uint32_t> aiVertexPositionIndexArrayOffsets(iNumClusters);
    std::vector<uint32_t> aiNumVertexPositionIndices(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        //DEBUG_PRINTF("cluster %d vertex position offset: %d vertex position index offset: %d\n",
        //    iCluster,
        //    iCurrVertexPositionDataOffset,
        //    iCurrVertexPositionIndexDataOffset);

        aiNumVertexPositionComponents[iCluster] = static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);
        aiVertexPositionComponentArrayOffsets[iCluster] = iCurrVertexPositionDataOffset;
        iCurrVertexPositionDataOffset += static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);

        aiNumVertexPositionIndices[iCluster] = static_cast<uint32_t>(aaiVertexPositionIndices[iCluster].size());
        aiVertexPositionIndexArrayOffsets[iCluster] = iCurrVertexPositionIndexDataOffset;
        iCurrVertexPositionIndexDataOffset += static_cast<uint32_t>(aaiVertexPositionIndices[iCluster].size());
    }

    // allocate device memory
    uint32_t* paaiRetAdjacentEdgeClusters;
    hipMalloc(&paaiRetAdjacentEdgeClusters, iNumClusters * iNumClusters * sizeof(uint32_t));
    hipMemset(paaiRetAdjacentEdgeClusters, 0xff, iNumClusters * iNumClusters * sizeof(uint32_t));

    uint32_t* paiRetNumAdjacentEdgeClusters;
    hipMalloc(&paiRetNumAdjacentEdgeClusters, iNumClusters * sizeof(uint32_t));
    hipMemset(paiRetNumAdjacentEdgeClusters, 0, iNumClusters * sizeof(uint32_t));

    uint32_t* paiNumVertexPositionComponents;
    hipMalloc(&paiNumVertexPositionComponents, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiNumVertexPositionComponents,
        aiNumVertexPositionComponents.data(),
        aiNumVertexPositionComponents.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* paiVertexPositionComponentOffsets;
    hipMalloc(&paiVertexPositionComponentOffsets, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiVertexPositionComponentOffsets,
        aiVertexPositionComponentArrayOffsets.data(),
        aiVertexPositionComponentArrayOffsets.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* paiVertexPositionIndexOffsets;
    hipMalloc(&paiVertexPositionIndexOffsets, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiVertexPositionIndexOffsets,
        aiVertexPositionIndexArrayOffsets.data(),
        aiVertexPositionIndexArrayOffsets.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* paiNumVertexPositionIndices;
    hipMalloc(&paiNumVertexPositionIndices, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiNumVertexPositionIndices,
        aiNumVertexPositionIndices.data(),
        aiNumVertexPositionIndices.size() * sizeof(int),
        hipMemcpyHostToDevice);


    // copy vertex positions
    float* pafTotalClusterVertexPositions;
    hipMalloc(&pafTotalClusterVertexPositions, iCurrVertexPositionDataOffset * sizeof(float));
    uint32_t iArrayIndexOffset = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        hipMemcpy(
            pafTotalClusterVertexPositions + iArrayIndexOffset,
            aaVertexPositions[iCluster].data(),
            aaVertexPositions[iCluster].size() * sizeof(float) * 3,
            hipMemcpyHostToDevice);
        iArrayIndexOffset += static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);
    }

    // copy vertex indices
    uint32_t* paaiVertexPositionIndices;
    hipMalloc(&paaiVertexPositionIndices, iCurrVertexPositionIndexDataOffset * sizeof(uint32_t));
    iArrayIndexOffset = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        hipMemcpy(
            paaiVertexPositionIndices + iArrayIndexOffset,
            aaiVertexPositionIndices[iCluster].data(),
            aaiVertexPositionIndices[iCluster].size() * sizeof(uint32_t),
            hipMemcpyHostToDevice);

        iArrayIndexOffset += static_cast<uint32_t>(aaiVertexPositionIndices[iCluster].size());
    }

    float* pafClusterMinMaxCenterRadius;
    hipMalloc(&pafClusterMinMaxCenterRadius, iNumClusters * 10 * sizeof(float));

    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumClusters) / float(WORKGROUP_SIZE)));
    getClusterBounds<<<iNumBlocks, WORKGROUP_SIZE>>>(
        pafClusterMinMaxCenterRadius,
        pafTotalClusterVertexPositions,
        paiVertexPositionComponentOffsets,
        paiNumVertexPositionComponents,
        iNumClusters);

    std::vector<float> afClusterMinMaxCenterRadiusCPU(iNumClusters * 10);
    hipMemcpy(
        afClusterMinMaxCenterRadiusCPU.data(),
        pafClusterMinMaxCenterRadius,
        afClusterMinMaxCenterRadiusCPU.size() * sizeof(float),
        hipMemcpyDeviceToHost);

    std::vector<vec3> aMinBounds(iNumClusters);
    std::vector<vec3> aMaxBounds(iNumClusters);
    std::vector<vec3> aCenter(iNumClusters);
    std::vector<float> afRadius(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        aMinBounds[iCluster].x = afClusterMinMaxCenterRadiusCPU[iCluster * 10];
        aMinBounds[iCluster].y = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 1];
        aMinBounds[iCluster].z = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 2];

        aMaxBounds[iCluster].x = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 3];
        aMaxBounds[iCluster].y = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 4];
        aMaxBounds[iCluster].z = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 5];

        aCenter[iCluster].x = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 6];
        aCenter[iCluster].y = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 7];
        aCenter[iCluster].z = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 8];

        afRadius[iCluster] = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 9];
    }

    float* pafRetDistances;
    hipMalloc(&pafRetDistances, iNumClusters * iNumClusters * sizeof(float));
    hipMemset(pafRetDistances, 0, iNumClusters * iNumClusters * sizeof(float));

    float* pafClusterCenters;
    hipMalloc(&pafClusterCenters, iNumClusters * 3 * sizeof(float));
    hipMemcpy(
        pafClusterCenters,
        aCenter.data(),
        iNumClusters * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    getClusterDistances<<<iNumBlocks, WORKGROUP_SIZE>>>(
        pafRetDistances,
        pafClusterCenters,
        iNumClusters);

    std::vector<float> afRetDistancesCPU(iNumClusters * iNumClusters);
    hipMemcpy(
        afRetDistancesCPU.data(),
        pafRetDistances,
        iNumClusters * iNumClusters * sizeof(float),
        hipMemcpyDeviceToHost);

    struct ClusterDistanceInfo
    {
        uint32_t        miCluster;
        float           mfDistance;
    };

    std::vector<std::vector< ClusterDistanceInfo>> aaClusterDistanceInfo(iNumClusters);

    std::vector<std::vector<float>> aafDistances(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        aaClusterDistanceInfo[iCluster].resize(iNumClusters);

        aafDistances[iCluster].resize(iNumClusters);
        for(uint32_t iCheckCluster = 0; iCheckCluster < iNumClusters; iCheckCluster++)
        {
            uint32_t iIndex = iCluster * iNumClusters + iCheckCluster;
            aafDistances[iCluster][iCheckCluster] = afRetDistancesCPU[iIndex];

            aaClusterDistanceInfo[iCluster][iCheckCluster].miCluster = iCheckCluster;
            aaClusterDistanceInfo[iCluster][iCheckCluster].mfDistance = aafDistances[iCluster][iCheckCluster];
        }

        std::sort(
            aaClusterDistanceInfo[iCluster].begin(),
            aaClusterDistanceInfo[iCluster].end(),
            [](ClusterDistanceInfo const& checkInfo0, ClusterDistanceInfo const& checkInfo1)
            {
                return checkInfo0.mfDistance < checkInfo1.mfDistance;
            }
        );
    }

    aaiSortedAdjacentEdgeClusters.resize(iNumClusters);
    for(uint32_t i = 0; i < iNumClusters; i++)
    {
        aaiSortedAdjacentEdgeClusters[i].resize(iNumClusters);
        for(uint32_t j = 0; j < iNumClusters; j++)
        {
            aaiSortedAdjacentEdgeClusters[i][j] = aaClusterDistanceInfo[i][j].miCluster;
        }
    }

    hipFree(pafClusterMinMaxCenterRadius);
    hipFree(pafRetDistances);
    hipFree(pafClusterCenters);
    hipFree(paaiRetAdjacentEdgeClusters);
    hipFree(paiRetNumAdjacentEdgeClusters);
    hipFree(paiNumVertexPositionComponents);
    hipFree(paiVertexPositionComponentOffsets);
    hipFree(paiVertexPositionIndexOffsets);
    hipFree(paiNumVertexPositionIndices);
    hipFree(pafTotalClusterVertexPositions);
    hipFree(paaiVertexPositionIndices);

    auto end = std::chrono::high_resolution_clock::now();
    uint64_t iSeconds = std::chrono::duration_cast<std::chrono::seconds>(end - start).count();
    DEBUG_PRINTF("*** took %lld seconds for getSortedEdgeAdjacentClustersCUDA to finish ***\n",
        iSeconds);
}

/*
**  TODO: finish this
*/
void buildClusterEdgeAdjacencyCUDA(
    std::vector<std::vector<uint32_t>>& aaiAdjacentEdgeClusters,
    std::vector<std::vector<vec3>> const& aaVertexPositions,
    std::vector<std::vector<uint32_t>> const& aaiVertexPositionIndices)
{
    DEBUG_PRINTF("*** start buildClusterEdgeAdjacencyCUDA ***\n");
    auto start = std::chrono::high_resolution_clock::now();

    uint32_t iCurrVertexPositionIndexDataOffset = 0;
    uint32_t iCurrVertexPositionDataOffset = 0;
    uint32_t iNumClusters = static_cast<uint32_t>(aaVertexPositions.size());
    std::vector<uint32_t> aiNumVertexPositionComponents(iNumClusters);
    std::vector<uint32_t> aiVertexPositionComponentArrayOffsets(iNumClusters);
    std::vector<uint32_t> aiVertexPositionIndexArrayOffsets(iNumClusters);
    std::vector<uint32_t> aiNumVertexPositionIndices(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        //DEBUG_PRINTF("cluster %d vertex position offset: %d vertex position index offset: %d\n",
        //    iCluster,
        //    iCurrVertexPositionDataOffset,
        //    iCurrVertexPositionIndexDataOffset);

        aiNumVertexPositionComponents[iCluster] = static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);
        aiVertexPositionComponentArrayOffsets[iCluster] = iCurrVertexPositionDataOffset;
        iCurrVertexPositionDataOffset += static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);

        aiNumVertexPositionIndices[iCluster] = static_cast<uint32_t>(aaiVertexPositionIndices[iCluster].size());
        aiVertexPositionIndexArrayOffsets[iCluster] = iCurrVertexPositionIndexDataOffset;
        iCurrVertexPositionIndexDataOffset += static_cast<uint32_t>(aaiVertexPositionIndices[iCluster].size());
    }

    // allocate device memory
    uint32_t* paaiRetAdjacentEdgeClusters;
    hipMalloc(&paaiRetAdjacentEdgeClusters, iNumClusters * iNumClusters * sizeof(uint32_t));
    hipMemset(paaiRetAdjacentEdgeClusters, 0xff, iNumClusters * iNumClusters * sizeof(uint32_t));

    uint32_t* paiRetNumAdjacentEdgeClusters;
    hipMalloc(&paiRetNumAdjacentEdgeClusters, iNumClusters * sizeof(uint32_t));
    hipMemset(paiRetNumAdjacentEdgeClusters, 0, iNumClusters * sizeof(uint32_t));

    uint32_t* paiNumVertexPositionComponents;
    hipMalloc(&paiNumVertexPositionComponents, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiNumVertexPositionComponents,
        aiNumVertexPositionComponents.data(),
        aiNumVertexPositionComponents.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* paiVertexPositionComponentOffsets;
    hipMalloc(&paiVertexPositionComponentOffsets, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiVertexPositionComponentOffsets,
        aiVertexPositionComponentArrayOffsets.data(),
        aiVertexPositionComponentArrayOffsets.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* paiVertexPositionIndexOffsets;
    hipMalloc(&paiVertexPositionIndexOffsets, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiVertexPositionIndexOffsets,
        aiVertexPositionIndexArrayOffsets.data(),
        aiVertexPositionIndexArrayOffsets.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* paiNumVertexPositionIndices;
    hipMalloc(&paiNumVertexPositionIndices, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiNumVertexPositionIndices,
        aiNumVertexPositionIndices.data(),
        aiNumVertexPositionIndices.size() * sizeof(int),
        hipMemcpyHostToDevice);

       

    // copy vertex positions
    float* pafTotalClusterVertexPositions;
    hipMalloc(&pafTotalClusterVertexPositions, iCurrVertexPositionDataOffset * sizeof(float));
    uint32_t iArrayIndexOffset = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        hipMemcpy(
            pafTotalClusterVertexPositions + iArrayIndexOffset,
            aaVertexPositions[iCluster].data(),
            aaVertexPositions[iCluster].size() * sizeof(float) * 3,
            hipMemcpyHostToDevice);
        iArrayIndexOffset += static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);
    }

    // copy vertex indices
    uint32_t* paaiVertexPositionIndices;
    hipMalloc(&paaiVertexPositionIndices, iCurrVertexPositionIndexDataOffset * sizeof(uint32_t));
    iArrayIndexOffset = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        hipMemcpy(
            paaiVertexPositionIndices + iArrayIndexOffset,
            aaiVertexPositionIndices[iCluster].data(),
            aaiVertexPositionIndices[iCluster].size() * sizeof(uint32_t),
            hipMemcpyHostToDevice);

        iArrayIndexOffset += static_cast<uint32_t>(aaiVertexPositionIndices[iCluster].size());
    }

    float* pafClusterMinMaxCenterRadius;
    hipMalloc(&pafClusterMinMaxCenterRadius, iNumClusters * 10 * sizeof(float));

    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumClusters) / float(WORKGROUP_SIZE)));
    getClusterBounds<<<iNumBlocks, WORKGROUP_SIZE>>>(
        pafClusterMinMaxCenterRadius,
        pafTotalClusterVertexPositions,
        paiVertexPositionComponentOffsets,
        paiNumVertexPositionComponents,
        iNumClusters);
    
    std::vector<float> afClusterMinMaxCenterRadiusCPU(iNumClusters * 10);
    hipMemcpy(
        afClusterMinMaxCenterRadiusCPU.data(),
        pafClusterMinMaxCenterRadius,
        afClusterMinMaxCenterRadiusCPU.size() * sizeof(float),
        hipMemcpyDeviceToHost);

    std::vector<vec3> aMinBounds(iNumClusters);
    std::vector<vec3> aMaxBounds(iNumClusters);
    std::vector<vec3> aCenter(iNumClusters);
    std::vector<float> afRadius(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        aMinBounds[iCluster].x = afClusterMinMaxCenterRadiusCPU[iCluster * 10];
        aMinBounds[iCluster].y = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 1];
        aMinBounds[iCluster].z = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 2];

        aMaxBounds[iCluster].x = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 3];
        aMaxBounds[iCluster].y = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 4];
        aMaxBounds[iCluster].z = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 5];

        aCenter[iCluster].x = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 6];
        aCenter[iCluster].y = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 7];
        aCenter[iCluster].z = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 8];

        afRadius[iCluster] = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 9];
    }
    
    float* pafRetDistances;
    hipMalloc(&pafRetDistances, iNumClusters * iNumClusters * sizeof(float));
    hipMemset(pafRetDistances, 0, iNumClusters * iNumClusters * sizeof(float));
    
    float* pafClusterCenters;
    hipMalloc(&pafClusterCenters, iNumClusters * 3 * sizeof(float));
    hipMemcpy(
        pafClusterCenters,
        aCenter.data(),
        iNumClusters * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    getClusterDistances<<<iNumBlocks,WORKGROUP_SIZE>>>(
        pafRetDistances,
        pafClusterCenters,
        iNumClusters);

    std::vector<float> afRetDistancesCPU(iNumClusters * iNumClusters);
    hipMemcpy(
        afRetDistancesCPU.data(),
        pafRetDistances,
        iNumClusters * iNumClusters * sizeof(float),
        hipMemcpyDeviceToHost);

    struct ClusterDistanceInfo
    {
        uint32_t        miCluster;
        float           mfDistance;
    };

    std::vector<std::vector< ClusterDistanceInfo>> aaClusterDistanceInfo(iNumClusters);

    std::vector<std::vector<float>> aafDistances(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        aaClusterDistanceInfo[iCluster].resize(iNumClusters);

        aafDistances[iCluster].resize(iNumClusters);
        for(uint32_t iCheckCluster = 0; iCheckCluster < iNumClusters; iCheckCluster++)
        {
            uint32_t iIndex = iCluster * iNumClusters + iCheckCluster;
            aafDistances[iCluster][iCheckCluster] = afRetDistancesCPU[iIndex];

            aaClusterDistanceInfo[iCluster][iCheckCluster].miCluster = iCheckCluster;
            aaClusterDistanceInfo[iCluster][iCheckCluster].mfDistance = aafDistances[iCluster][iCheckCluster];
        }

        std::sort(
            aaClusterDistanceInfo[iCluster].begin(),
            aaClusterDistanceInfo[iCluster].end(),
            [](ClusterDistanceInfo const& checkInfo0, ClusterDistanceInfo const& checkInfo1)
            {
                return checkInfo0.mfDistance < checkInfo1.mfDistance;
            }
        );
    }

    std::vector<uint32_t> aiSortedClusters(iNumClusters * iNumClusters);
    for(uint32_t i = 0; i < iNumClusters; i++)
    {
        for(uint32_t j = 0; j < iNumClusters; j++)
        {
            uint32_t iIndex = i * iNumClusters + j;
            aiSortedClusters[iIndex] = aaClusterDistanceInfo[i][j].miCluster;
        }
    }

    hipFree(pafClusterMinMaxCenterRadius);
    hipFree(pafRetDistances);
    hipFree(pafClusterCenters);

    uint32_t* paiDistanceSortedClusterID;
    hipMalloc(&paiDistanceSortedClusterID, iNumClusters* iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiDistanceSortedClusterID,
        aiSortedClusters.data(),
        aiSortedClusters.size() * sizeof(uint32_t),
        hipMemcpyHostToDevice);

    buildClusterEdgeAdjacency<<<iNumBlocks, WORKGROUP_SIZE>>>(
        paaiRetAdjacentEdgeClusters,
        paiRetNumAdjacentEdgeClusters,
        pafTotalClusterVertexPositions,
        paaiVertexPositionIndices,
        paiNumVertexPositionComponents,
        paiNumVertexPositionIndices,
        paiVertexPositionComponentOffsets,
        paiVertexPositionIndexOffsets,
        paiDistanceSortedClusterID,
        iNumClusters);

    std::vector<uint32_t> aaiRetAdjacentEdgeClustersCPU(iNumClusters * iNumClusters);
    hipMemcpy(
        aaiRetAdjacentEdgeClustersCPU.data(),
        paaiRetAdjacentEdgeClusters,
        iNumClusters * iNumClusters * sizeof(uint32_t),
        hipMemcpyDeviceToHost);

    std::vector<uint32_t> aiRetNumAdjacentEdgeClustersCPU(iNumClusters);
    hipMemcpy(
        aiRetNumAdjacentEdgeClustersCPU.data(),
        paiRetNumAdjacentEdgeClusters,
        iNumClusters * sizeof(int),
        hipMemcpyDeviceToHost);

    aaiAdjacentEdgeClusters.resize(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        for(uint32_t i = 0; i < iNumClusters; i++)
        {
            uint32_t iIndex = iCluster * iNumClusters + i;
            if(aaiRetAdjacentEdgeClustersCPU[iIndex] != UINT32_MAX)
            {
                aaiAdjacentEdgeClusters[iCluster].push_back(i);
            }
        }
    }

    hipFree(paaiRetAdjacentEdgeClusters);
    hipFree(paiRetNumAdjacentEdgeClusters);
    hipFree(paiNumVertexPositionComponents);
    hipFree(paiVertexPositionComponentOffsets);
    hipFree(paiVertexPositionIndexOffsets);
    hipFree(paiNumVertexPositionIndices);
    hipFree(pafTotalClusterVertexPositions);
    hipFree(paaiVertexPositionIndices);
    hipFree(paiDistanceSortedClusterID);

    auto end = std::chrono::high_resolution_clock::now();
    uint64_t iSeconds = std::chrono::duration_cast<std::chrono::seconds>(end - start).count();
    DEBUG_PRINTF("*** took %lld seconds for buildClusterEdgeAdjacencyCUDA to finish ***\n",
        iSeconds);

}

/*
**
*/
void buildClusterEdgeAdjacencyCUDA2(
    std::vector<std::vector<std::pair<uint32_t, uint32_t>>>& aaiAdjacentEdgeClusters,
    std::vector<std::vector<vec3>> const& aaVertexPositions,
    std::vector<std::vector<uint32_t>> const& aaiVertexPositionIndices)
{
    DEBUG_PRINTF("*** start buildClusterEdgeAdjacencyCUDA ***\n");
    auto start = std::chrono::high_resolution_clock::now();

    uint32_t iCurrVertexPositionIndexDataOffset = 0;
    uint32_t iCurrVertexPositionDataOffset = 0;
    uint32_t iNumClusters = static_cast<uint32_t>(aaVertexPositions.size());
    std::vector<uint32_t> aiNumVertexPositionComponents(iNumClusters);
    std::vector<uint32_t> aiVertexPositionComponentArrayOffsets(iNumClusters);
    std::vector<uint32_t> aiVertexPositionIndexArrayOffsets(iNumClusters);
    std::vector<uint32_t> aiNumVertexPositionIndices(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        //DEBUG_PRINTF("cluster %d vertex position offset: %d vertex position index offset: %d\n",
        //    iCluster,
        //    iCurrVertexPositionDataOffset,
        //    iCurrVertexPositionIndexDataOffset);

        aiNumVertexPositionComponents[iCluster] = static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);
        aiVertexPositionComponentArrayOffsets[iCluster] = iCurrVertexPositionDataOffset;
        iCurrVertexPositionDataOffset += static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);

        aiNumVertexPositionIndices[iCluster] = static_cast<uint32_t>(aaiVertexPositionIndices[iCluster].size());
        aiVertexPositionIndexArrayOffsets[iCluster] = iCurrVertexPositionIndexDataOffset;
        iCurrVertexPositionIndexDataOffset += static_cast<uint32_t>(aaiVertexPositionIndices[iCluster].size());
    }

    // allocate device memory
    uint32_t* paaiRetAdjacentEdgeClusters;
    hipMalloc(&paaiRetAdjacentEdgeClusters, iNumClusters * iNumClusters * sizeof(uint32_t));
    hipMemset(paaiRetAdjacentEdgeClusters, 0xff, iNumClusters * iNumClusters * sizeof(uint32_t));

    uint32_t* paiRetNumAdjacentEdgeClusters;
    hipMalloc(&paiRetNumAdjacentEdgeClusters, iNumClusters * sizeof(uint32_t));
    hipMemset(paiRetNumAdjacentEdgeClusters, 0, iNumClusters * sizeof(uint32_t));

    uint32_t* paiNumVertexPositionComponents;
    hipMalloc(&paiNumVertexPositionComponents, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiNumVertexPositionComponents,
        aiNumVertexPositionComponents.data(),
        aiNumVertexPositionComponents.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* paiVertexPositionComponentOffsets;
    hipMalloc(&paiVertexPositionComponentOffsets, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiVertexPositionComponentOffsets,
        aiVertexPositionComponentArrayOffsets.data(),
        aiVertexPositionComponentArrayOffsets.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* paiVertexPositionIndexOffsets;
    hipMalloc(&paiVertexPositionIndexOffsets, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiVertexPositionIndexOffsets,
        aiVertexPositionIndexArrayOffsets.data(),
        aiVertexPositionIndexArrayOffsets.size() * sizeof(int),
        hipMemcpyHostToDevice);

    uint32_t* paiNumVertexPositionIndices;
    hipMalloc(&paiNumVertexPositionIndices, iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiNumVertexPositionIndices,
        aiNumVertexPositionIndices.data(),
        aiNumVertexPositionIndices.size() * sizeof(int),
        hipMemcpyHostToDevice);



    // copy vertex positions
    float* pafTotalClusterVertexPositions;
    hipMalloc(&pafTotalClusterVertexPositions, iCurrVertexPositionDataOffset * sizeof(float));
    uint32_t iArrayIndexOffset = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        hipMemcpy(
            pafTotalClusterVertexPositions + iArrayIndexOffset,
            aaVertexPositions[iCluster].data(),
            aaVertexPositions[iCluster].size() * sizeof(float) * 3,
            hipMemcpyHostToDevice);
        iArrayIndexOffset += static_cast<uint32_t>(aaVertexPositions[iCluster].size() * 3);
    }

    // copy vertex indices
    uint32_t* paaiVertexPositionIndices;
    hipMalloc(&paaiVertexPositionIndices, iCurrVertexPositionIndexDataOffset * sizeof(uint32_t));
    iArrayIndexOffset = 0;
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        hipMemcpy(
            paaiVertexPositionIndices + iArrayIndexOffset,
            aaiVertexPositionIndices[iCluster].data(),
            aaiVertexPositionIndices[iCluster].size() * sizeof(uint32_t),
            hipMemcpyHostToDevice);

        iArrayIndexOffset += static_cast<uint32_t>(aaiVertexPositionIndices[iCluster].size());
    }

    float* pafClusterMinMaxCenterRadius;
    hipMalloc(&pafClusterMinMaxCenterRadius, iNumClusters * 10 * sizeof(float));

    uint32_t iNumBlocks = static_cast<uint32_t>(ceilf(static_cast<float>(iNumClusters) / float(WORKGROUP_SIZE)));
    getClusterBounds << <iNumBlocks, WORKGROUP_SIZE >> > (
        pafClusterMinMaxCenterRadius,
        pafTotalClusterVertexPositions,
        paiVertexPositionComponentOffsets,
        paiNumVertexPositionComponents,
        iNumClusters);

    std::vector<float> afClusterMinMaxCenterRadiusCPU(iNumClusters * 10);
    hipMemcpy(
        afClusterMinMaxCenterRadiusCPU.data(),
        pafClusterMinMaxCenterRadius,
        afClusterMinMaxCenterRadiusCPU.size() * sizeof(float),
        hipMemcpyDeviceToHost);

    std::vector<vec3> aMinBounds(iNumClusters);
    std::vector<vec3> aMaxBounds(iNumClusters);
    std::vector<vec3> aCenter(iNumClusters);
    std::vector<float> afRadius(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        aMinBounds[iCluster].x = afClusterMinMaxCenterRadiusCPU[iCluster * 10];
        aMinBounds[iCluster].y = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 1];
        aMinBounds[iCluster].z = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 2];

        aMaxBounds[iCluster].x = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 3];
        aMaxBounds[iCluster].y = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 4];
        aMaxBounds[iCluster].z = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 5];

        aCenter[iCluster].x = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 6];
        aCenter[iCluster].y = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 7];
        aCenter[iCluster].z = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 8];

        afRadius[iCluster] = afClusterMinMaxCenterRadiusCPU[iCluster * 10 + 9];
    }

    float* pafRetDistances;
    hipMalloc(&pafRetDistances, iNumClusters * iNumClusters * sizeof(float));
    hipMemset(pafRetDistances, 0, iNumClusters * iNumClusters * sizeof(float));

    float* pafClusterCenters;
    hipMalloc(&pafClusterCenters, iNumClusters * 3 * sizeof(float));
    hipMemcpy(
        pafClusterCenters,
        aCenter.data(),
        iNumClusters * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    getClusterDistances << <iNumBlocks, WORKGROUP_SIZE >> > (
        pafRetDistances,
        pafClusterCenters,
        iNumClusters);

    std::vector<float> afRetDistancesCPU(iNumClusters * iNumClusters);
    hipMemcpy(
        afRetDistancesCPU.data(),
        pafRetDistances,
        iNumClusters * iNumClusters * sizeof(float),
        hipMemcpyDeviceToHost);

    struct ClusterDistanceInfo
    {
        uint32_t        miCluster;
        float           mfDistance;
    };

    std::vector<std::vector< ClusterDistanceInfo>> aaClusterDistanceInfo(iNumClusters);

    std::vector<std::vector<float>> aafDistances(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        aaClusterDistanceInfo[iCluster].resize(iNumClusters);

        aafDistances[iCluster].resize(iNumClusters);
        for(uint32_t iCheckCluster = 0; iCheckCluster < iNumClusters; iCheckCluster++)
        {
            uint32_t iIndex = iCluster * iNumClusters + iCheckCluster;
            aafDistances[iCluster][iCheckCluster] = afRetDistancesCPU[iIndex];

            aaClusterDistanceInfo[iCluster][iCheckCluster].miCluster = iCheckCluster;
            aaClusterDistanceInfo[iCluster][iCheckCluster].mfDistance = aafDistances[iCluster][iCheckCluster];
        }

        std::sort(
            aaClusterDistanceInfo[iCluster].begin(),
            aaClusterDistanceInfo[iCluster].end(),
            [](ClusterDistanceInfo const& checkInfo0, ClusterDistanceInfo const& checkInfo1)
            {
                return checkInfo0.mfDistance < checkInfo1.mfDistance;
            }
        );
    }

    std::vector<uint32_t> aiSortedClusters(iNumClusters * iNumClusters);
    for(uint32_t i = 0; i < iNumClusters; i++)
    {
        for(uint32_t j = 0; j < iNumClusters; j++)
        {
            uint32_t iIndex = i * iNumClusters + j;
            aiSortedClusters[iIndex] = aaClusterDistanceInfo[i][j].miCluster;
        }
    }

    hipFree(pafClusterMinMaxCenterRadius);
    hipFree(pafRetDistances);
    hipFree(pafClusterCenters);

    uint32_t* paiDistanceSortedClusterID;
    hipMalloc(&paiDistanceSortedClusterID, iNumClusters * iNumClusters * sizeof(uint32_t));
    hipMemcpy(
        paiDistanceSortedClusterID,
        aiSortedClusters.data(),
        aiSortedClusters.size() * sizeof(uint32_t),
        hipMemcpyHostToDevice);

    buildClusterEdgeAdjacency2<<<iNumBlocks, WORKGROUP_SIZE>>>(
        paaiRetAdjacentEdgeClusters,
        paiRetNumAdjacentEdgeClusters,
        pafTotalClusterVertexPositions,
        paaiVertexPositionIndices,
        paiNumVertexPositionComponents,
        paiNumVertexPositionIndices,
        paiVertexPositionComponentOffsets,
        paiVertexPositionIndexOffsets,
        paiDistanceSortedClusterID,
        iNumClusters);

    std::vector<uint32_t> aaiRetAdjacentEdgeClustersCPU(iNumClusters * iNumClusters);
    hipMemcpy(
        aaiRetAdjacentEdgeClustersCPU.data(),
        paaiRetAdjacentEdgeClusters,
        iNumClusters * iNumClusters * sizeof(uint32_t),
        hipMemcpyDeviceToHost);

    std::vector<uint32_t> aiRetNumAdjacentEdgeClustersCPU(iNumClusters);
    hipMemcpy(
        aiRetNumAdjacentEdgeClustersCPU.data(),
        paiRetNumAdjacentEdgeClusters,
        iNumClusters * sizeof(int),
        hipMemcpyDeviceToHost);

    aaiAdjacentEdgeClusters.resize(iNumClusters);
    for(uint32_t iCluster = 0; iCluster < iNumClusters; iCluster++)
    {
        for(uint32_t i = 0; i < iNumClusters; i++)
        {
            uint32_t iIndex = iCluster * iNumClusters + i;
            if(aaiRetAdjacentEdgeClustersCPU[iIndex] != UINT32_MAX)
            {
                aaiAdjacentEdgeClusters[iCluster].push_back(std::make_pair(i, aaiRetAdjacentEdgeClustersCPU[iIndex]));
            }
        }
    }

    hipFree(paaiRetAdjacentEdgeClusters);
    hipFree(paiRetNumAdjacentEdgeClusters);
    hipFree(paiNumVertexPositionComponents);
    hipFree(paiVertexPositionComponentOffsets);
    hipFree(paiVertexPositionIndexOffsets);
    hipFree(paiNumVertexPositionIndices);
    hipFree(pafTotalClusterVertexPositions);
    hipFree(paaiVertexPositionIndices);
    hipFree(paiDistanceSortedClusterID);

    auto end = std::chrono::high_resolution_clock::now();
    uint64_t iSeconds = std::chrono::duration_cast<std::chrono::seconds>(end - start).count();
    DEBUG_PRINTF("*** took %lld seconds for buildClusterEdgeAdjacencyCUDA2 to finish ***\n",
        iSeconds);
}