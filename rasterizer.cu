#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>

#define uint32_t unsigned int
#define int32_t int

inline __host__ __device__ float3 operator + (float3& a, float3& b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

inline __host__ __device__ float3 operator - (float3& a, float3& b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __host__ __device__ float3 operator * (float3& a, float3& b)
{
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

inline __host__ __device__ float3 operator / (float3& a, float3& b)
{
    return make_float3(a.x / b.x, a.y / b.y, a.z / b.z);
}

inline __host__ __device__ float3 operator * (float3& a, float& b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

inline __host__ __device__ float3 fminf(float3 a, float3 b)
{
    return make_float3(fminf(a.x, b.x), fminf(a.y, b.y), fminf(a.z, b.z));
}

inline __host__ __device__ float3 fmaxf(float3 a, float3 b)
{
    return make_float3(fmaxf(a.x, b.x), fmaxf(a.y, b.y), fmaxf(a.z, b.z));
}

inline __device__ __host__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}

inline __device__ __host__ float dot(float3& a, float3& b)
{
    return (a.x * b.x + a.y * b.y + a.z * b.y);
}

inline __device__ __host__ float3 barycentric(
    float3& p, 
    float3& a, 
    float3& b, 
    float3& c)
{
    float3 v0 = b - a, v1 = c - a, v2 = p - a;
    float fD00 = dot(v0, v0);
    float fD01 = dot(v0, v1);
    float fD11 = dot(v1, v1);
    float fD20 = dot(v2, v0);
    float fD21 = dot(v2, v1);
    float fDenom = fD00 * fD11 - fD01 * fD01;
    float fV = (fD11 * fD20 - fD01 * fD21) / fDenom;
    float fW = (fD00 * fD21 - fD01 * fD20) / fDenom;
    float fU = 1.0f - fV - fW;

    return make_float3(fU, fV, fW);
}

__device__ __forceinline__ float atomicMaxFloat(float* addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
        __uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

    return old;
}

/*
**
*/
__device__ 
void _rasterizeTriangle(
    float* pafPositionBuffer,
    float* pafNormalBuffer,
    float* pafDepthBuffer,
    float* pafColorBuffer,
    float3 pos0,
    float3 pos1,
    float3 pos2,
    float3 normal0,
    float3 normal1,
    float3 normal2,
    float3 color0,
    float3 color1,
    float3 color2,
    uint32_t iBufferWidth,
    uint32_t iBufferHeight,
    uint32_t iTriangle)
{
    float3 screenDimension = make_float3(float(iBufferWidth), float(iBufferHeight), 0.0f);

    float3 screenCoord0 = pos0 * screenDimension;
    float3 screenCoord1 = pos1 * screenDimension;
    float3 screenCoord2 = pos2 * screenDimension;

    float3 minScreenPos = fminf(screenCoord0, fminf(screenCoord1, screenCoord2));
    float3 maxScreenPos = fmaxf(screenCoord0, fmaxf(screenCoord1, screenCoord2));

    screenCoord0.x = (screenCoord0.x == minScreenPos.x) ? floorf(screenCoord0.x) : screenCoord0.x;
    screenCoord0.y = (screenCoord0.y == minScreenPos.y) ? floorf(screenCoord0.y) : screenCoord0.y;
    screenCoord0.x = (screenCoord0.x == maxScreenPos.x) ? ceilf(screenCoord0.x) : screenCoord0.x;
    screenCoord0.y = (screenCoord0.y == maxScreenPos.y) ? ceilf(screenCoord0.y) : screenCoord0.y;

    screenCoord1.x = (screenCoord1.x == minScreenPos.x) ? floorf(screenCoord1.x) : screenCoord1.x;
    screenCoord1.y = (screenCoord1.y == minScreenPos.y) ? floorf(screenCoord1.y) : screenCoord1.y;
    screenCoord1.x = (screenCoord1.x == maxScreenPos.x) ? ceilf(screenCoord1.x) : screenCoord1.x;
    screenCoord1.y = (screenCoord1.y == maxScreenPos.y) ? ceilf(screenCoord1.y) : screenCoord1.y;

    screenCoord2.x = (screenCoord2.x == minScreenPos.x) ? floorf(screenCoord2.x) : screenCoord2.x;
    screenCoord2.y = (screenCoord2.y == minScreenPos.y) ? floorf(screenCoord2.y) : screenCoord2.y;
    screenCoord2.x = (screenCoord2.x == maxScreenPos.x) ? ceilf(screenCoord2.x) : screenCoord2.x;
    screenCoord2.y = (screenCoord2.y == maxScreenPos.y) ? ceilf(screenCoord2.y) : screenCoord2.y;

    uint32_t iScreenX0 = clamp(static_cast<uint32_t>(floorf(minScreenPos.x)), 0, iBufferWidth - 1);
    uint32_t iScreenX1 = clamp(static_cast<uint32_t>(ceilf(maxScreenPos.x)), 0, iBufferWidth - 1);

    uint32_t iScreenY0 = clamp(static_cast<uint32_t>(floorf(minScreenPos.y)), 0, iBufferHeight - 1);
    uint32_t iScreenY1 = clamp(static_cast<uint32_t>(ceilf(maxScreenPos.y)), 0, iBufferHeight - 1);

    // compute barycentric coordinate within the face 2d boundary to fetch the clipspace position and normal
    for(uint32_t iY = iScreenY0; iY <= iScreenY1; iY++)
    {
        for(uint32_t iX = iScreenX0; iX <= iScreenX1; iX++)
        {
            float3 currPos = make_float3(float(iX), float(iY), 0.0f);
            float3 barycentricCoord = barycentric(
                currPos,
                screenCoord0,
                screenCoord1,
                screenCoord2);

            if(barycentricCoord.x >= 0.0f && barycentricCoord.y >= 0.0f && barycentricCoord.x + barycentricCoord.y <= 1.0f)
            {
                // check depth buffer (larger than incoming depth -> replace)
                float3 currPos = pos0 * barycentricCoord.x + pos1 * barycentricCoord.y + pos2 * barycentricCoord.z;
                float3 normal = normal0 * barycentricCoord.x + normal1 * barycentricCoord.y + normal2 * barycentricCoord.z;
                uint32_t iIndex = iY * iBufferWidth + iX;
                if(pafDepthBuffer[iIndex] > currPos.z)
                {
                    pafPositionBuffer[iIndex * 3] = currPos.x;
                    pafPositionBuffer[iIndex * 3 + 1] = currPos.y;
                    pafPositionBuffer[iIndex * 3 + 2] = currPos.z;

                    pafNormalBuffer[iIndex * 3] = normal.x;
                    pafNormalBuffer[iIndex * 3 + 1] = normal.y;
                    pafNormalBuffer[iIndex * 3 + 2] = normal.z;

                    pafDepthBuffer[iIndex] = currPos.z;

                    pafColorBuffer[iIndex * 3] = color0.x;
                    pafColorBuffer[iIndex * 3 + 1] = color0.y;
                    pafColorBuffer[iIndex * 3 + 2] = color0.z;
                }
            }
        }
    }
}

/*
**
*/
__global__
void _rasterizeMesh(
    float* paPositionBuffer,
    float* paNormalBuffer,
    float* paDepthBuffer,
    float* paColorBuffer,
    float* paVertexPositions,
    float* paVertexNormals,
    float* paVertexUVs,
    uint32_t* paiVertexPositionIndices,
    uint32_t* paiVertexNormalIndices,
    uint32_t* paiVertexUVIndices,
    uint32_t* paiCountBuffers,
    uint32_t iBufferWidth,
    uint32_t iBufferHeight)
{
    uint32_t iNumTriangleVertices = paiCountBuffers[0];
    uint32_t iNumTriangles = iNumTriangleVertices / 3;

    uint32_t iTriangle = blockIdx.x * 512 + threadIdx.x;
    if(iTriangle >= iNumTriangles)
    {
        return;
    }

    uint32_t iIndex = iTriangle * 3;
    uint32_t iPos0 = paiVertexPositionIndices[iIndex] * 3;
    uint32_t iPos1 = paiVertexPositionIndices[iIndex + 1] * 3;
    uint32_t iPos2 = paiVertexPositionIndices[iIndex + 2] * 3;

    float3 pos0 = make_float3(
        paVertexPositions[iPos0], 
        paVertexPositions[iPos0 + 1], 
        paVertexPositions[iPos0 + 2]);
    
    float3 pos1 = make_float3(
        paVertexPositions[iPos1],
        paVertexPositions[iPos1 + 1],
        paVertexPositions[iPos1 + 2]);

    float3 pos2 = make_float3(
        paVertexPositions[iPos2],
        paVertexPositions[iPos2 + 1],
        paVertexPositions[iPos2 + 2]);

    uint32_t iNorm0 = paiVertexNormalIndices[iIndex] * 3;
    uint32_t iNorm1 = paiVertexNormalIndices[iIndex + 1] * 3;
    uint32_t iNorm2 = paiVertexNormalIndices[iIndex + 2] * 3;

    float3 normal0 = make_float3(
        paVertexNormals[iNorm0],
        paVertexNormals[iNorm0 + 1],
        paVertexNormals[iNorm0 + 2]);

    float3 normal1 = make_float3(
        paVertexNormals[iNorm1],
        paVertexNormals[iNorm1 + 1],
        paVertexNormals[iNorm1 + 2]);

    float3 normal2 = make_float3(
        paVertexNormals[iNorm2],
        paVertexNormals[iNorm2 + 1],
        paVertexNormals[iNorm2 + 2]);

    float3 color0 = make_float3(1.0f, 1.0f, 1.0f);
    float3 color1 = make_float3(1.0f, 1.0f, 1.0f);
    float3 color2 = make_float3(1.0f, 1.0f, 1.0f);

    _rasterizeTriangle(
        paPositionBuffer,
        paNormalBuffer,
        paDepthBuffer,
        paColorBuffer,
        pos0,
        pos1,
        pos2,
        normal0,
        normal1,
        normal2,
        color0,
        color1,
        color2,
        iBufferWidth,
        iBufferHeight,
        iTriangle);

}

/*
**
*/
__global__
void _rasterizeMesh2(
    float* paPositionBuffer,
    float* paNormalBuffer,
    float* paDepthBuffer,
    float* paColorBuffer,
    float* paVertexPositions,
    float* paVertexNormals,
    float* paVertexColors,
    uint32_t* paiCountBuffers,
    uint32_t iBufferWidth,
    uint32_t iBufferHeight)
{
    uint32_t iNumTriangleVertices = paiCountBuffers[0];
    uint32_t iNumTriangles = iNumTriangleVertices / 3;

    uint32_t iTriangle = blockIdx.x * 512 + threadIdx.x;
    if(iTriangle >= iNumTriangles)
    {
        return;
    }

    uint32_t iPos0 = iTriangle * 9;
    uint32_t iPos1 = iPos0 + 3;
    uint32_t iPos2 = iPos1 + 3;

    float3 pos0 = make_float3(
        paVertexPositions[iPos0],
        paVertexPositions[iPos0 + 1],
        paVertexPositions[iPos0 + 2]);

    float3 pos1 = make_float3(
        paVertexPositions[iPos1],
        paVertexPositions[iPos1 + 1],
        paVertexPositions[iPos1 + 2]);

    float3 pos2 = make_float3(
        paVertexPositions[iPos2],
        paVertexPositions[iPos2 + 1],
        paVertexPositions[iPos2 + 2]);

    uint32_t iNorm0 = iPos0;
    uint32_t iNorm1 = iPos1;
    uint32_t iNorm2 = iPos2;

    float3 normal0 = make_float3(
        paVertexNormals[iNorm0],
        paVertexNormals[iNorm0 + 1],
        paVertexNormals[iNorm0 + 2]);

    float3 normal1 = make_float3(
        paVertexNormals[iNorm1],
        paVertexNormals[iNorm1 + 1],
        paVertexNormals[iNorm1 + 2]);

    float3 normal2 = make_float3(
        paVertexNormals[iNorm2],
        paVertexNormals[iNorm2 + 1],
        paVertexNormals[iNorm2 + 2]);

    float3 color0 = make_float3(
        paVertexColors[iPos0],
        paVertexColors[iPos0 + 1],
        paVertexColors[iPos0 + 2]);

    float3 color1 = make_float3(
        paVertexColors[iPos1],
        paVertexColors[iPos1 + 1],
        paVertexColors[iPos1 + 2]);

    float3 color2 = make_float3(
        paVertexColors[iPos2],
        paVertexColors[iPos2 + 1],
        paVertexColors[iPos2 + 2]);

    _rasterizeTriangle(
        paPositionBuffer,
        paNormalBuffer,
        paDepthBuffer,
        paColorBuffer,
        pos0,
        pos1,
        pos2,
        normal0,
        normal1,
        normal2,
        color0,
        color1,
        color2,
        iBufferWidth,
        iBufferHeight,
        iTriangle);

}

/*
**
*/
__global__
void compositeImage(
    float* paColorBuffer,
    float* paPositionBuffer,
    float* paNormalBuffer,
    float* paAlbedoBuffer,
    float* pLightDirection,
    uint32_t iImageWidth,
    uint32_t iImageHeight)
{
    uint32_t iPixel = blockIdx.x * 512 + threadIdx.x;
    if(iPixel >= iImageWidth * iImageHeight)
    {
        return;
    }

    uint32_t iPixelIndex = iPixel * 3;
    float3 position = make_float3(paPositionBuffer[iPixelIndex], paPositionBuffer[iPixelIndex + 1], paPositionBuffer[iPixelIndex + 2]);
    float3 normal = make_float3(paNormalBuffer[iPixelIndex], paNormalBuffer[iPixelIndex + 1], paNormalBuffer[iPixelIndex + 2]);
    float3 color = make_float3(paAlbedoBuffer[iPixelIndex], paAlbedoBuffer[iPixelIndex + 1], paAlbedoBuffer[iPixelIndex + 2]);
    if(normal.x == 0.0f && normal.y == 0.0f && normal.z == 0.0f)
    {
        return;
    }

    float3 lightDirection = make_float3(pLightDirection[0], pLightDirection[1], pLightDirection[2]);
    
    float fAmbient = 0.0f;

    float fIntensity = max(dot(lightDirection, normal), 0.0f);
    paColorBuffer[iPixelIndex] =        fIntensity * color.x + fAmbient;
    paColorBuffer[iPixelIndex + 1] =    fIntensity * color.y + fAmbient;
    paColorBuffer[iPixelIndex + 2] =    fIntensity * color.z + fAmbient;
}

#undef uint32_t
#undef int32_t


#include "vec.h"
#include <vector>

#if 0
/*
**
*/
void rasterizeMeshCUDA(
    std::vector<vec4>& retColorBuffer,
    std::vector<vec3>& retNormalBuffer,
    std::vector<float>& retDepthBuffer,
    std::vector<vec3> const& aVertexPositions,
    std::vector<vec3> const& aVertexNormals,
    std::vector<vec2> const& aVertexUVs,
    std::vector<uint32_t> const& aiVertexPositionIndices,
    std::vector<uint32_t> const& aiVertexNormalIndices,
    std::vector<uint32_t> const& aiVertexUVIndices,
    std::vector<vec3> const& inputColorBuffer,
    std::vector<vec3> const& inputNormalBuffer, 
    std::vector<float> const& inputDepthBuffer)
{
    float* paVertexPositions;
    hipMalloc(
        &paVertexPositions,
        aVertexPositions.size() * 3 * sizeof(float));
    hipMemcpy(
        paVertexPositions,
        aVertexPositions.data(),
        aVertexPositions.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    float* paVertexNormals;
    hipMalloc(
        &paVertexNormals,
        aVertexNormals.size() * 3 * sizeof(float));
    hipMemcpy(
        paVertexNormals,
        aVertexNormals.data(),
        aVertexNormals.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    float* paVertexUVs;
    hipMalloc(
        &paVertexUVs,
        aVertexUVs.size() * 2 * sizeof(float));
    hipMemcpy(
        paVertexUVs,
        aVertexUVs.data(),
        aVertexUVs.size() * 2 * sizeof(float),
        hipMemcpyHostToDevice);

    uint32_t* paiVertexPositionIndices;
    hipMalloc(
        &paiVertexPositionIndices,
        aiVertexPositionIndices.size() * sizeof(uint32_t));
    hipMemcpy(
        paiVertexPositionIndices,
        aiVertexPositionIndices.data(),
        aiVertexPositionIndices.size() * sizeof(uint32_t),
        hipMemcpyHostToDevice);

    uint32_t* paiVertexNormalIndices;
    hipMalloc(
        &paiVertexNormalIndices,
        aiVertexNormalIndices.size() * sizeof(uint32_t));
    hipMemcpy(
        paiVertexNormalIndices,
        aiVertexNormalIndices.data(),
        aiVertexNormalIndices.size() * sizeof(uint32_t),
        hipMemcpyHostToDevice);

    uint32_t* paiVertexUVIndices;
    hipMalloc(
        &paiVertexUVIndices,
        aiVertexUVIndices.size() * sizeof(uint32_t));
    hipMemcpy(
        paiVertexUVIndices,
        aiVertexUVIndices.data(),
        aiVertexUVIndices.size() * sizeof(uint32_t),
        hipMemcpyHostToDevice);

    uint32_t const kiImageWidth = 1024;
    uint32_t const kiImageHeight = 1024;
    uint32_t const kiFormatSize = 3;

    float* paPositionBuffer;
    hipMalloc(
        &paPositionBuffer,
        kiImageWidth * kiImageHeight * kiFormatSize * sizeof(float));
    hipMemcpy(
        paPositionBuffer,
        inputColorBuffer.data(),
        kiImageWidth * kiImageHeight * kiFormatSize * sizeof(float),
        hipMemcpyHostToDevice);

    float* paNormalBuffer;
    hipMalloc(
        &paNormalBuffer,
        kiImageWidth * kiImageHeight * kiFormatSize * sizeof(float));
    hipMemcpy(
        paNormalBuffer,
        inputNormalBuffer.data(),
        kiImageWidth * kiImageHeight * kiFormatSize * sizeof(float),
        hipMemcpyHostToDevice);

    float* paDepthBuffer;
    hipMalloc(
        &paDepthBuffer,
        kiImageWidth * kiImageHeight * sizeof(float));
    hipMemcpy(
        paDepthBuffer,
        inputDepthBuffer.data(),
        kiImageWidth * kiImageHeight * sizeof(float),
        hipMemcpyHostToDevice);

    std::vector<uint32_t> aiCount(3);
    aiCount[0] = static_cast<uint32_t>(aiVertexPositionIndices.size());
    aiCount[1] = static_cast<uint32_t>(aiVertexNormalIndices.size());
    aiCount[2] = static_cast<uint32_t>(aiVertexUVIndices.size());

    uint32_t* paiCountBuffers;
    hipMalloc(
        &paiCountBuffers,
        64 * sizeof(uint32_t));
    hipMemcpy(
        paiCountBuffers,
        aiCount.data(),
        3 * sizeof(uint32_t),
        hipMemcpyHostToDevice);

    uint32_t iNumBlocks = max((aiCount[0] / 3) / 512, 1);
    _rasterizeMesh<<<iNumBlocks, 512>>>(
        paPositionBuffer,
        paNormalBuffer,
        paDepthBuffer,
        paColorBuffer,
        paVertexPositions,
        paVertexNormals,
        paVertexUVs,
        paiVertexPositionIndices,
        paiVertexNormalIndices,
        paiVertexUVIndices,
        paiCountBuffers,
        kiImageWidth,
        kiImageHeight);

    hipMemcpy(
        retColorBuffer.data(),
        paPositionBuffer,
        kiImageWidth * kiImageHeight * kiFormatSize * sizeof(float),
        hipMemcpyDeviceToHost);
    
    hipMemcpy(
        retNormalBuffer.data(),
        paNormalBuffer,
        kiImageWidth * kiImageHeight * kiFormatSize * sizeof(float),
        hipMemcpyDeviceToHost);

    hipMemcpy(
        retDepthBuffer.data(),
        paDepthBuffer,
        kiImageWidth * kiImageHeight * sizeof(float),
        hipMemcpyDeviceToHost);

    hipFree(paVertexPositions);
    hipFree(paVertexNormals);
    hipFree(paVertexUVs);
    hipFree(paiVertexPositionIndices);
    hipFree(paiVertexNormalIndices);
    hipFree(paiVertexUVIndices);

    hipFree(paiCountBuffers);
    hipFree(paPositionBuffer);
    hipFree(paNormalBuffer);
    hipFree(paDepthBuffer);
}
#endif // #if 0

/*
**
*/
void rasterizeMeshCUDA2(
    std::vector<vec3>& retLightIntensityBuffer,
    std::vector<vec3>& retPositionBuffer,
    std::vector<vec3>& retNormalBuffer,
    std::vector<float>& retDepthBuffer,
    std::vector<vec3>& retColorBuffer,
    std::vector<vec3> const& aVertexPositions,
    std::vector<vec3> const& aVertexNormals,
    std::vector<vec3> const& aVertexColors,
    uint32_t iImageWidth,
    uint32_t iImageHeight,
    uint32_t iImageFormatSize)
{
    float* paVertexPositions;
    hipMalloc(
        &paVertexPositions,
        aVertexPositions.size() * 3 * sizeof(float));
    hipMemcpy(
        paVertexPositions,
        aVertexPositions.data(),
        aVertexPositions.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    float* paVertexNormals;
    hipMalloc(
        &paVertexNormals,
        aVertexNormals.size() * 3 * sizeof(float));
    hipMemcpy(
        paVertexNormals,
        aVertexNormals.data(),
        aVertexNormals.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);

    float* paVertexColors;
    hipMalloc(
        &paVertexColors,
        aVertexColors.size() * 3 * sizeof(float));
    hipMemcpy(
        paVertexColors,
        aVertexColors.data(),
        aVertexColors.size() * 3 * sizeof(float),
        hipMemcpyHostToDevice);
    

    float* paPositionBuffer;
    hipMalloc(
        &paPositionBuffer,
        iImageWidth * iImageHeight * iImageFormatSize * sizeof(float));
    hipMemcpy(
        paPositionBuffer,
        retPositionBuffer.data(),
        iImageWidth * iImageHeight * iImageFormatSize * sizeof(float),
        hipMemcpyHostToDevice);

    float* paNormalBuffer;
    hipMalloc(
        &paNormalBuffer,
        iImageWidth * iImageHeight * iImageFormatSize * sizeof(float));
    hipMemcpy(
        paNormalBuffer,
        retNormalBuffer.data(),
        iImageWidth * iImageHeight * iImageFormatSize * sizeof(float),
        hipMemcpyHostToDevice);

    float* paDepthBuffer;
    hipMalloc(
        &paDepthBuffer,
        iImageWidth * iImageHeight * sizeof(float));
    hipMemcpy(
        paDepthBuffer,
        retDepthBuffer.data(),
        iImageWidth * iImageHeight * sizeof(float),
        hipMemcpyHostToDevice);

    float* paColorBuffer;
    hipMalloc(
        &paColorBuffer,
        iImageWidth * iImageHeight * iImageFormatSize * sizeof(float));
    hipMemcpy(
        paColorBuffer,
        retColorBuffer.data(),
        iImageWidth * iImageHeight * iImageFormatSize * sizeof(float),
        hipMemcpyHostToDevice);
    
    std::vector<uint32_t> aiCount(3);
    aiCount[0] = static_cast<uint32_t>(aVertexPositions.size());
    aiCount[1] = static_cast<uint32_t>(aVertexPositions.size());
    aiCount[2] = static_cast<uint32_t>(aVertexPositions.size());

    uint32_t* paiCountBuffers;
    hipMalloc(
        &paiCountBuffers,
        64 * sizeof(uint32_t));
    hipMemcpy(
        paiCountBuffers,
        aiCount.data(),
        3 * sizeof(uint32_t),
        hipMemcpyHostToDevice);

    uint32_t iNumBlocks = max(static_cast<uint32_t>(ceilf(float(aiCount[0] / 3) / 512.0f)), 1);
    _rasterizeMesh2<<<iNumBlocks, 512>>>(
        paPositionBuffer,
        paNormalBuffer,
        paDepthBuffer,
        paColorBuffer,
        paVertexPositions,
        paVertexNormals,
        paVertexColors,
        paiCountBuffers,
        iImageWidth,
        iImageHeight);

    
    vec3 lightDirection = normalize(vec3(1.0f, 1.0f, 1.0f));
    float* pLightDirection;
    hipMalloc(
        &pLightDirection,
        sizeof(float) * 4);
    hipMemcpy(
        pLightDirection,
        &lightDirection,
        sizeof(float) * 3,
        hipMemcpyHostToDevice);
    
    float* pLightOutputBuffer;
    hipMalloc(
        &pLightOutputBuffer,
        iImageWidth * iImageHeight * iImageFormatSize * sizeof(float));
    hipMemset(
        pLightOutputBuffer,
        0,
        iImageWidth* iImageHeight* iImageFormatSize * sizeof(float));
    
    iNumBlocks = max(static_cast<uint32_t>(ceilf(float(iImageWidth * iImageHeight) / 512.0f)), 1);
    compositeImage<<<iNumBlocks, 512>>>(
        pLightOutputBuffer,
        paPositionBuffer,
        paNormalBuffer,
        paColorBuffer,
        pLightDirection,
        iImageWidth,
        iImageHeight);

    hipMemcpy(
        retLightIntensityBuffer.data(),
        pLightOutputBuffer,
        iImageWidth* iImageHeight* iImageFormatSize * sizeof(float),
        hipMemcpyDeviceToHost);

    hipMemcpy(
        retPositionBuffer.data(),
        paPositionBuffer,
        iImageWidth * iImageHeight * iImageFormatSize * sizeof(float),
        hipMemcpyDeviceToHost);

    hipMemcpy(
        retNormalBuffer.data(),
        paNormalBuffer,
        iImageWidth * iImageHeight * iImageFormatSize * sizeof(float),
        hipMemcpyDeviceToHost);

    hipMemcpy(
        retDepthBuffer.data(),
        paDepthBuffer,
        iImageWidth * iImageHeight * sizeof(float),
        hipMemcpyDeviceToHost);

    hipMemcpy(
        retColorBuffer.data(),
        paColorBuffer,
        iImageWidth* iImageHeight* iImageFormatSize * sizeof(float),
        hipMemcpyDeviceToHost);

    hipFree(paVertexPositions);
    hipFree(paVertexNormals);
    hipFree(paVertexColors);

    hipFree(paiCountBuffers);
    hipFree(paPositionBuffer);
    hipFree(paNormalBuffer);
    hipFree(paDepthBuffer);
    hipFree(paColorBuffer);
    hipFree(pLightOutputBuffer);

    hipFree(pLightDirection);
    
}